#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include "grid.h"
#define MAX 4
#define N 10
void initCell(cell * c);
__global__ void compute2(returnResult * res, grid * g, path * p, location * l);
__device__ void computeIterative(returnResult * res, grid * g, path * p, location * baseLoc);
__device__ void add(grid ** base, grid ** last, grid * newList);
__device__ void cloneToGrid(grid * g, grid * g2);
__device__ void eliminateValue(cell **c, int row, int col, int max, int value);
__device__ int check(grid * g, int row, int col, int number);
__device__ grid * allocateGridDevice(int size);
__global__ void testIter(returnResult * res);
void printGrid(grid * g);
__device__ int pow2(int x);
__device__ grid * cloneGrid(grid * g);
char convert(int x);
void bar();
int foo(path * p);
__global__ void testIter(returnResult * res)
{
	grid *  g = allocateGridDevice(res->result[0]->size);
	grid ** temp = res->result;
	for(int i = 0; i < res->breaker; i++)
	{
		int index = i % res->size;
		cloneToGrid(g, temp[index]);
		temp[index]-> ok = '1';
	}
}
int main(void)
	{
		bar();
	}
void foobared()
	{
		int size = 10;
		grid ** result;
		int i;
		int last = 0; 
		int gridSize;
		returnResult * res;
		hipMallocManaged((void **) &res, 1);
		res->threads = 1;
		hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024 * 8);
		gridSize = 1057 * res->threads;
		hipMallocManaged((void **) &result, sizeof(grid *) * gridSize);
		for (i = 0; i < gridSize; i++)
				{
					result[i] = allocateGrid(size);
				}
		for(int breaker =1050; breaker < 1000000; breaker+=1000 )
		{
			printf("Starting %d\n", breaker);
			res->result = result;
			res->breaker = breaker;
			res->size = gridSize;
			testIter<<<1, res->threads>>>(res);
			hipDeviceSynchronize();
			for (i = 0; i < gridSize; i++)
				{		
					if (result[i]->ok == '1')
						{
						last = i;
					}
				}
			printf("Size %d Grid #%d", gridSize, last);
			printf("Grid #%d", 0);
			printGrid(result[0]);
			printf("Grid #%d", last);
			printGrid(result[last]);
		}
		
	}
void bar()
{
		int deviceCount;
		hipGetDeviceCount(&deviceCount);
		int device;
		for (device = 0; device < deviceCount; ++device)
			{
				hipDeviceProp_t deviceProp;
				hipGetDeviceProperties(&deviceProp, device);
				printf("Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor);
			} //	 - See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/#multi-device-system
		hipSetDevice(0);
		path ** p = scanChars();
		if (p != NULL)
			{
				foo(p[1]);
			}
	}
__global__ void compute2(returnResult * res, grid * g, path * p, location * l)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < 1)
			{
				//int x = blockIdx.x;
				//int y = threadIdx.x;
				computeIterative(res, g, p, l);
			}
	}
__device__ void computeIterative(returnResult * res, grid * g, path * p, location * baseLoc)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int xx = res->size/res->threads * idx;
		grid ** result = &res->result[xx];
		int gridSize = res->size/res->threads;
		int breaker = 0;
		int bmax = 2;
		int printcount = 0;
		location * loc = &baseLoc[idx];
		location * freeHead = NULL;
		int i = 0;
		if (p->direction == LEFT) //Do UP/DOWN
			{
				loc->nx = loc->x;
				loc->ny = 0;
			}
		else
			{
				loc->nx = 0;
				loc->ny = loc->y;
			}
		location * temp;
		int checkValue;
		int value;
		int z;
		int done = 0;
		//int idx = blockIdx.x * blockDim.x + threadIdx.x;
		//int base = idx * MAX *3 + recCount;
		grid * currentGrid = allocateGridDevice(g->size);
		cloneToGrid(g, currentGrid);
		loc->currentG = allocateGridDevice(g->size);
		loc->p = p;
		loc->next = NULL;
		//recCount = recCount +3 ;
		//int set = 0;
		int x, y;
		int count = 0;
		while (done == 0)
			{
				breaker++;
				cloneToGrid(loc->currentG, currentGrid);
				x = loc->x;
				y = loc->y;

				p = loc->p;
				int lasty = y;
				int lastx = x;
				value = p->letters[0];
				checkValue = check(currentGrid, x, y, value);
				if (checkValue == 0)
					{
						currentGrid->cells[x][y].value = value;
						eliminateValue(currentGrid->cells, x, y, currentGrid->size, value);
						int direction;
						if (p->direction == LEFT) //Do UP/DOWN
							{
								z = loc->ny;
								loc->ny++;
							}
						else
							{
								z = loc->nx;
								loc->nx++;
							}
						checkValue = 0;
						if (p->direction == LEFT) //Do UP/DOWN
							direction = y > z ? -1 : 1;
						else
							direction = x > z ? -1 : 1;
						for (int offset = 0; offset < 3 && checkValue == 0; offset++)
							{
								value = p->letters[offset + 1];
								if (p->direction == LEFT) //Do UP/DOWN
									lasty = (z + (offset * direction) + currentGrid->size) % currentGrid->size;
								else
									lastx = (z + (offset * direction) + currentGrid->size) % currentGrid->size;
								checkValue |= check(currentGrid, lastx, lasty, value);
								if (checkValue == 0)
									{
										currentGrid->cells[lastx][lasty].value = value;
										eliminateValue(currentGrid->cells, lastx, lasty, currentGrid->size, value);
									}
							}
						//if(checkValue == 0)
						//{
						//	printGrid(currentGrid);
						//}

					}
				if (checkValue == 0 && count == MAX)
					{
						i++;
						//if (printcount < gridSize-1)
							{
								int offset = printcount % gridSize;
								//printf("breaker@@ = %d offset %d\n", breaker, offset);
								cloneToGrid(currentGrid, result[offset]);
								result[offset]->ok = '1';
								//prinotGrid(currentGrid);
								printcount++;
							}
						//else
							{
						//		done = 1;
							}
					}
				if (p->direction == LEFT)
						{
							z = loc->ny;
						}
					else
						{
							z = loc->nx;
						}
				if (checkValue == 0 && count < MAX && z < g->size) //rec value
					{
						//cloneToGrid(currentGrid, res[base]
						
						if (p->next != NULL)
							{
								if (freeHead == NULL)
									{
										temp = (location *) malloc(sizeof(location));
										temp->currentG = allocateGridDevice(g->size);
										printf("Allocated Block\n");
									}
								else
									{
										temp = freeHead;
										if (freeHead->next != NULL)
											{
												freeHead = freeHead->next;
											}
									}
								temp->x = lastx;
								temp->y = lasty;
								if (p->next->direction == LEFT) //Do UP/DOWN
									{
										
										temp->nx = temp->x;
										temp->ny = 0;	
									}
								else
									{
										temp->nx = 0;
										temp->ny = temp->y;
									}
								//printf("Next x=%d y=%d nx=%d ny=%d\n",temp->x,temp->y,temp->nx,temp->ny);
								temp->p = p->next;
								cloneToGrid(currentGrid, temp->currentG);
								temp->next = loc;
								loc = temp;
								count++;
								printf("Push count=%d loc x%d y%d nx%d ny%d \n", count,loc->x, loc->y, loc->nx, loc->ny);
							}
					}
				else 
					{
						
						if (z == g->size) //pop off the list
							{
								if (loc->next == NULL)
									{
										done = 1;
									}
								else
									{
										temp = loc->next;
										if (freeHead == NULL)
											{
												freeHead = loc;
												freeHead->next = NULL;
											}
										else
											{
												
												loc->next = freeHead;
												freeHead = loc;
												//freeHead->next = loc;
												//freeTail = loc;
												//freeTail->next = NULL;
												
											}
										loc = temp;
										count--;
										printf("Pop count=%d loc x%d y%d nx%d ny%d \n", count,loc->x, loc->y, loc->nx, loc->ny);
									}
							}
					}
				if (bmax == breaker)
					{
						printf("Breaker %d PrintCount  %d\n", breaker, printcount);
						bmax *= 1.2;
					}
				if (breaker == res->breaker)
					{
						done = 1;
						printf("Breaker Max hit!");
					}
			}
		printf("The total is %d breaker %d\n", i, breaker);

	}

int foo(path * p)
	{
		returnResult * res;
		hipMallocManaged((void **) &res, 1);
		hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024 * 8); //See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#heap-memory-allocation
		int nBYn = N * N;
		int size = N;
		grid * g = allocateGrid(size);
		
		grid ** result;
		int i;
		int last = 0;
		location * larray;
		hipMallocManaged((void **) &larray, sizeof(location) * nBYn);
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{
						int offset = row * N + col;
						larray[offset].x = row;
						larray[offset].y = col;
					}
			}
		printPath(p);
		
		res->threads = 1;
		int gridSize = 1057 * res->threads;
		hipMallocManaged((void **) &result, sizeof(grid *) * gridSize);
		for (i = 0; i < gridSize; i++)
			{
				result[i] = allocateGrid(size);
			}
		//for(int breaker =100000; breaker < 10000000; breaker+=100000)
		//int breaker = 100000000;
		//for(int gridSize = 1000; gridSize < 1057; gridSize++)
		{
			clock_t begin = clock();
			int breaker = 100000;
			printf("Starting %d\n", breaker);
			res->result = result;
			res->breaker = breaker;
			res->size = gridSize;
			compute2<<<1, res->threads>>>(res, g, p, larray);
			hipDeviceSynchronize();
			for (i = 0; i < gridSize; i++)
				{		
					if (result[i]->ok == '1')
						{
						last = i;
						//printf("Grid #%d", i);
						//printGrid(result[i]);
					}
				}
			printf("Size %d Grid #%d", gridSize, last);
			printf("Grid #%d", 0);
			printGrid(result[0]);
			printf("Grid #%d", last);
			printGrid(result[last]);
			//printf("Done %d\n", breaker);
			clock_t end = clock();
			double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
			printf("Time spent %lf iterations %d\n",time_spent,breaker);
		}
		/*i = 0;
		 for (int row = 0; row < N; row++)
		 {
		 for (int col = 0; col < N; col++)
		 {

		 for(int j = 0; j <MAX * 3; j+=3)
		 {
		 int idx = (row * size + col) * MAX*3 +j;
		 if (result[idx]->ok == '1')
		 {
		 printf("(%d,%d,%d)\n", row, col, j);
		 printGrid(result[idx]);
		 }
		 i++;
		 }
		 }
		 }
		 */
		/*for( int i = 0; i < nBYn * MAX * 3; i++)
		 {
		 if (result[i]->ok == '1')
		 {
		 //printf("(%d,%d,%d)\n", row, col, j);
		 puts("");
		 printGrid(result[i]);
		 }
		 }
		 */
		//hipFree(array);
		return 0;
	}

__device__ int pow2(int x)
	{
		int sum = 1;
		if (x == 0)
			{
				sum = 1;
			}
		else
			{
				for (int i = 0; i < x; i++)
					{
						sum = sum * 2;
					}
			}
		return sum;
	}
__device__ void add(grid ** base, grid ** last, grid * newList)
	{
		if (newList != NULL)
			{
				if (*base == NULL)
					{
						*base = newList;
						*last = *base;
					}
				else
					{
						(*last)->next = newList;
					}
				while ((*last)->next != NULL)
					{
						*last = (*last)->next;
					}
			}
	}

char convert(int x)
	{
		char res = 'a';
		if (x >= 0)
			{
				//int amount = log2((double) x) + (int) res;
				int amount = int(x) + (int) res;

				res = (char) amount;
			}
		else
			{
				res = ' ';
			}
		return res;
	}

__device__ void cloneToGrid(grid * g, grid * g2)
	{
		g2->size = g->size;
		g2->ok = g->ok;
		for (int row = 0; row < g->size; row++)
			{
				for (int col = 0; col < g->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
	}

grid * allocateGrid(int size)
	{
		grid * g2 = NULL;
		hipMallocManaged((void **) &g2, sizeof(grid));
		g2->size = size;
		cell ** cells;
		hipMallocManaged((void **) &cells, size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				hipMallocManaged((void **) &cells[i], size * sizeof(cell));
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}

__device__ grid * allocateGridDevice(int size)
	{
		grid * g2 = NULL;
		g2 = (grid *) malloc(sizeof(grid));
		g2->size = size;
		cell ** cells;
		cells = (cell **) malloc(size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				cells[i] = (cell*) malloc(size * sizeof(cell));
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}

__device__ grid * cloneGrid(grid * g)
	{
		grid * g2 = (grid *) malloc(sizeof(grid));
		if (g2 != NULL)
			{
				g2->size = g->size;
				cell * array = (cell *) malloc(g->size * g2->size * sizeof(cell));
				cell ** cells = (cell **) malloc(g2->size * sizeof(cell *));
				for (int i = 0; i < g->size; i++)
					{
						cells[i] = &array[i * g2->size];
					}
				g2->cells = cells;
				for (int row = 0; row < g2->size; row++)
					{
						for (int col = 0; col < g2->size; col++)
							{
								g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
								g2->cells[row][col].value = g->cells[row][col].value;
							}
					}
				g2->next = NULL;
				g2->ok = '0';
			}
		return g2;
	}

__device__ void eliminateValue(cell **c, int row, int col, int max, int value)
	{
		int mask = pow2(value);
		for (int r1 = 0; r1 < max; r1++)
			{
				if (r1 != row)
					{
						c[r1][col].bitmap |= mask;
					}
			}
		for (int c1 = 0; c1 < max; c1++)
			{
				if (c1 != col)
					{
						c[row][c1].bitmap |= mask;
					}
			}
	}

__device__ int check(grid * g, int row, int col, int number)
	{
		int result;
		cell * c = &g->cells[row][col];
		if (c->value >= 0 && c->value != number)
			{
				result = 1;
			}
		else
			{
				int mask = pow2(number);
				int bits = c->bitmap;
				result = (mask & bits);
			}
		return result;
	}

void printGrid(grid * g)
	{
		int n = g->size;
		//printf("X=%d Y=%d %c\n", x, y, g->ok);
		printf("-- Grid -- \n");
		for (int row = 0; row < n; row++)
			{
				printf("%01d# ", row);
				for (int col = 0; col < n; col++)
					{
						cell c = g->cells[row][col];
						//printf("[%02d][%02d]%02X ",row, col, c[row * N + col].bitmap);
						int value = c.value;
						char printC = ' ';
						if (value < 0)
							{
								value = c.bitmap;
								printf(" %03X%c", value, printC);
							}
						else
							{
								//
								printC = convert(value);
								value = 0;
								printf("  %c  ", printC);
							}

					}
				printf("\n");
			}
	}
