#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include "grid.h"
#define N 10
int allocated = 0;
void initCell(cell * c);
gridResult * getGrids(path ** p, int MAX, int size);
__global__ void compute2(returnResult * res, grid * g, path ** pathList, location * l);
__global__ void compute3(returnResult * res, grid ** g, path ** pathlist, location * l);
__device__ void computeIterative(returnResult * res, grid * g, path ** pathList, location * baseLoc);
__device__ void add(grid ** base, grid ** last, grid * newList);
__device__ void cloneToGrid(grid * g, grid * g2);
__device__ void eliminateValue(cell **c, int row, int col, int max, int value);
__device__ int check(grid * g, int row, int col, int number);
__device__ grid * allocateGridDevice(int size);
__device__ int updateLocation(location * loc, path * p, int size);
void printGrid(grid * g);
__device__ int pow2(int x);
__device__ grid * cloneGrid(grid * g);
char convert(int x);
int foo(path ** p, int MAX, int breaker);
void processGrids(gridResult * grids, path ** path,int MAX, int size);


int main(int argc, char ** argv)
	{
		int MAX;
		int device;
		int breaker = 100000 * 10 * 10 * 10;
		int deviceCount;
		hipGetDeviceCount(&deviceCount);
		for (device = 0; device < deviceCount; ++device)
			{
				hipDeviceProp_t deviceProp;
				hipGetDeviceProperties(&deviceProp, device);
				printf("Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor);
			} //	 - See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/#multi-device-system
		if(argc == 3)
		{
			MAX = atoi(argv[1]);
			device = atoi(argv[2]);
		}
		else
		{
			printf("ARGS = MAX DEV\n");
			MAX = 5 * 2;
		}
		printf("Starting on device %d MAX %d breaker %d\n", device, MAX, breaker);
		hipSetDevice(device);
		path ** p = scanChars();
		if (p != NULL)
			{
				//foo(&p[0],MAX, breaker);
				gridResult * grids = getGrids(p,0,N);
				p[0] = p[0]->next;
				processGrids(grids, p,MAX, N);
			}
	}
void processGrids(gridResult * grids, path ** p,int MAX, int size)
{

	returnResult * res;
	hipMallocManaged((void **) &res, 1);
	grid * g = allocateGrid(size);
	grid ** result;
	location * larray;
	hipMallocManaged((void **) &larray, sizeof(location) * grids->size);
	for(int i = 0; i < grids->size; i++)
	{
		larray[i].x = grids->grids[i]->x;
		larray[i].y = grids->grids[i]->y;
		larray[i].full = PART;
	}
	res->threads = grids->size;
	int base = 512;
	int blocks = (grids->size % base)+1;
	int gridSize = 1;
	int amount = gridSize * sizeof(grid *);
	printf("Allocated Bytes %d\n", amount);
	hipMallocManaged((void **) &result, amount);
	for (int i = 0; i < gridSize; i++)
		{
			result[i] = allocateGrid(size);
		}
	amount = res->threads * sizeof(grid *) * (MAX + 1);
	printf("Allocated Bytes for GStack %d\n", amount);
	hipMallocManaged((void **) &res->gridStack, amount);
	for (int i = 0; i < res->threads * (MAX + 1); i++)
		{
			res->gridStack[i] = allocateGrid(size);
		}
	amount = sizeof(location) * (MAX + 1) * res->threads;
	printf("Allocated Bytes for LStack %d\n", amount);
	hipMallocManaged((void **) &res->locationStack, amount);
	res->result = result;
	res->size = gridSize;
	res->MAX = MAX;
	clock_t begin = clock();
	compute3<<<blocks, base>>>(res, grids->grids, p, larray);
	hipDeviceSynchronize();
	clock_t end = clock();
	double time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
	printf("Time spent %lf\n", time_spent);
	int last = 0;
	for (int i = 0; i < gridSize; i++)
			{
				if (result[i]->ok == '1')
						{
							last = i;
							printf("Grid #%d\n", i);
							printGrid(result[i]);
						}
			}
	printf("Size %d Grid #%d\n", gridSize, last);
}


gridResult * getGrids(path ** p, int MAX, int size)
{
	returnResult * res;
	hipMallocManaged((void **) &res, 1);
	grid * g = allocateGrid(size);
	grid ** result;
	location * larray;
	hipMallocManaged((void **) &larray, sizeof(location));
	larray[0].x = 0;
	larray[0].y = 0;
	larray[0].full = FULL;
	res->threads = 1;
	int gridSize = 1000;
	int amount = gridSize * sizeof(grid *);
	printf("Allocated Bytes %d\n", amount);
	hipMallocManaged((void **) &result, amount);
	for (int i = 0; i < gridSize; i++)
		{
			result[i] = allocateGrid(size);
		}
	amount = res->threads * sizeof(grid *) * (MAX + 1);
	printf("Allocated Bytes for GStack %d\n", amount);
	hipMallocManaged((void **) &res->gridStack, amount);
	for (int i = 0; i < res->threads * (MAX + 1); i++)
		{
			res->gridStack[i] = allocateGrid(size);
		}
	amount = sizeof(location) * (MAX + 1) * res->threads;
	printf("Allocated Bytes for LStack %d\n", amount);
	hipMallocManaged((void **) &res->locationStack, amount);
	res->result = result;
	res->size = gridSize;
	res->MAX = MAX;
	clock_t begin = clock();
	compute2<<<1, res->threads>>>(res, g, p, larray);
	hipDeviceSynchronize();
	clock_t end = clock();
	double time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
	printf("Time spent %lf\n", time_spent);
	int last = 0;
	for (int i = 0; i < gridSize; i++)
			{
				if (result[i]->ok == '1')
						{
							last = i;
							//printf("Grid #%d\n", i);
							//printGrid(result[i]);
						}
			}
	gridResult* grids = (gridResult *)malloc(sizeof(gridResult));
	grids->grids = result;
	grids->size = last;
	printf("Size %d Grid #%d\n", gridSize, last);
	return grids;
}


int foo(path ** p, int MAX, int breaker)
	{
		returnResult * res;
		hipMallocManaged((void **) &res, 1);
		//	hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024 * 8); //See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#heap-memory-allocation
		int nBYn = N * N;
		int size = N;
		grid * g = allocateGrid(size);

		grid ** result;
		int i;
		int last = 0;
		location * larray;
		hipMallocManaged((void **) &larray, sizeof(location) * nBYn);
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{
						int offset = row * N + col;
						larray[offset].x = row;
						larray[offset].y = col;
						larray[offset].full = PART;
					}
			}
		larray[0].full = PART;
		printPath(p[0]);
		printPath(p[1]);
		printPath(p[2]);
		printPath(p[3]);
		res->threads = 100;
		int gridSize = 1 * res->threads;
		int amount = gridSize * sizeof(grid *);
		printf("Allocated Bytes %d\n", amount);
		hipMallocManaged((void **) &result, amount);
		for (i = 0; i < gridSize; i++)
			{
				result[i] = allocateGrid(size);
			}
		amount = res->threads * sizeof(grid *) * (MAX + 1);
		printf("Allocated Bytes for GStack %d\n", amount);
		hipMallocManaged((void **) &res->gridStack, amount);
		for (i = 0; i < res->threads * (MAX + 1); i++)
			{
				res->gridStack[i] = allocateGrid(size);
			}
		amount = sizeof(location) * (MAX + 1) * res->threads;
		printf("Allocated Bytes for LStack %d\n", amount);
		hipMallocManaged((void **) &res->locationStack, amount);
		//for(int breaker =100000; breaker < 10000000; breaker+=100000)
			{

				
				printf("Starting %d\n", breaker);
				res->result = result;
				//res->breaker = breaker;
				res->size = gridSize;
				res->MAX = MAX;
				clock_t begin = clock();
				compute2<<<1, res->threads>>>(res, g, p, larray);
				//compute2<<<10, 10>>>(res, g, p, larray);
				hipDeviceSynchronize();
				clock_t end = clock();
				double time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
				printf("Time spent %lf iteration Max %d\n", time_spent, breaker);
				for (i = 0; i < gridSize; i++)
					{
						if (result[i]->ok == '1')
							{
								last = i;
								printf("Grid #%d\n", i);
								printGrid(result[i]);
							}
					}
				printf("Size %d Grid #%d", gridSize, last);
				/*printf("Grid #%d", 0);
				 printGrid(result[0]);
				 printf("Grid #%d", last);
				 printGrid(result[last]);*/
				//printf("Done %d\n", breaker);
			}
		return 0;
	}


__global__ void compute2(returnResult * res, grid * g, path ** pathlist, location * l)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < res->threads)
			{
				computeIterative(res, g, pathlist, l);
			}
	}
__global__ void compute3(returnResult * res, grid ** g, path ** pathlist, location * l)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < res->threads)
			{
				computeIterative(res, g[idx], pathlist, l);
			}
	}
__device__ void computeIterative(returnResult * res, grid * g, path ** pathList, location * baseLoc)
	{
		int baseIndex = 0;
		path * p = pathList[baseIndex]; //first path
		int MAX = res->MAX;
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int xx = res->size / res->threads * idx;
		int index = idx * (MAX + 1);
		grid ** gridStack = &res->gridStack[index];
		location * locStack = &res->locationStack[index];
		//printf("Index %d xx = %d gridIndex\n", idx, xx, index);
		grid ** result = &res->result[xx];
		int gridSize = res->size / res->threads;
		int breaker = 0;
		int bmax = 2;
		int printcount = 0;
		int count = 0;
		//location * loc = &baseLoc[idx];
		location * loc = &locStack[count];
		//copy location data
		loc->x = baseLoc[idx].x;
		loc->y = baseLoc[idx].y;
		loc->full = baseLoc[idx].full;

		//location * freeHead = NULL;
		int i = 0;
		int checkValue;
		int value;
		int done = 0;
		location * temp;
		if (p->direction == LEFT) //Do UP/DOWN
			{
				loc->nx = loc->x;
				loc->ny = 0;
			}
		else
			{
				loc->nx = 0;
				loc->ny = loc->y;
			}

		grid * currentGrid = allocateGridDevice(g->size);
		//grid * currentGrid = gridStack[count];
		cloneToGrid(g, currentGrid);
		cloneToGrid(currentGrid, gridStack[count]);
		//loc->currentG = gridStack[count];
		loc->p = p;
		loc->next = NULL;
		int pop;

		while (done == 0)
			{
				loc = &locStack[count];
				breaker++;
				pop = 0;
				cloneToGrid(gridStack[count], currentGrid);
				//currentGrid = gridStack[count];
				p = loc->p;
				int lasty = loc->y;
				int lastx = loc->x;
				//	printf("Count=%d loc x%d y%d nx%d ny%d \n", count,loc->x, loc->y, loc->nx, loc->ny);
				value = p->letters[0];
				checkValue = check(currentGrid, loc->x, loc->y, value);
				if (checkValue == 0)
					{
						currentGrid->cells[loc->x][loc->y].value = value;
						eliminateValue(currentGrid->cells, loc->x, loc->y, currentGrid->size, value);
						int direction;
						checkValue = 0;
						if (p->direction == LEFT) //Do UP/DOWN
							direction = loc->y > loc->ny ? -1 : 1;
						else
							direction = loc->x > loc->nx ? -1 : 1;
						for (int offset = 0; offset < 3 && checkValue == 0; offset++)
							{
								value = p->letters[offset + 1];
								if (p->direction == LEFT) //Do UP/DOWN
									lasty = (loc->ny + (offset * direction) + currentGrid->size) % currentGrid->size;
								else
									lastx = (loc->nx + (offset * direction) + currentGrid->size) % currentGrid->size;
								checkValue |= check(currentGrid, lastx, lasty, value);
								if (checkValue == 0)
									{
										currentGrid->cells[lastx][lasty].value = value;
										eliminateValue(currentGrid->cells, lastx, lasty, currentGrid->size, value);
									}
							}
					}
				if (checkValue == 0 && count == MAX)
					{
						i++;
						int offset = printcount % gridSize;
						//printf("breaker@@ = %d offset %d\n", breaker, offset);
						cloneToGrid(currentGrid, result[offset]);
						result[offset]->ok = '1';
						result[offset]->x = lastx;
						result[offset]->y = lasty;
						printcount++;
						//done = 1;
					}
				pop = updateLocation(loc, p, g->size);
				if (checkValue == 0 && count < MAX && pop == 0) //rec value
					{
						uint8_t type = PART;
						path * nextLoc = NULL;
						if (p->next != NULL)
							{
								nextLoc = p->next;
							}
						else
							{
								baseIndex++;
								if (pathList[baseIndex] != NULL)
									{
										nextLoc = pathList[baseIndex];
										type = FULL;
									}
							}
						if (nextLoc != NULL)
							{
								count++;
								temp = &locStack[count];
								temp->full = type;
								temp->x = lastx;
								temp->y = lasty;
								if (nextLoc->direction == LEFT) //Do UP/DOWN
									{
										temp->nx = temp->x;
										temp->ny = 0;
									}
								else
									{
										temp->nx = 0;
										temp->ny = temp->y;
									}
								temp->p = nextLoc;
								cloneToGrid(currentGrid, gridStack[count]);
								//	printf("Push count=%d loc x%d y%d nx%d ny%d \n", count,loc->x, loc->y, loc->nx, loc->ny);
							}
					}
				else
					{
						if (pop == 1) //pop off the list
							{
								if (loc->full == FULL)
									{
										baseIndex--;
										if (baseIndex < 0)
											{
												done = 1;
											}
									}
								count--;
								if (count < 0)
									{
										done = 1;
									}
							}
					}
				if (bmax == breaker)
					{
						//printf("Breaker %d PrintCount  %d\n", breaker, printcount);
						bmax *= 1.2;
					}
			/*	if (breaker == res->breaker)
					{
						done = 1;
						printf("Breaker Max hit!");
					}*/
			}
		printf("The total is %d breaker %d\n", i, breaker);
	}



__device__ int updateLocation(location * loc, path * p, int size)
	{
		int pop = 0;
		if (loc->full == PART)
					{
						if (p->direction == LEFT)
							{
								loc->ny++;
								if (loc->ny >= size)
									pop = 1;
							}
						else
							{
								loc->nx++;
								if (loc->nx >= size)
									pop = 1;
							}
					}
				else
					{
						if (p->direction == LEFT)
							{
								loc->ny++;
								if (loc->ny >= size)
									{
										loc->ny = 0;
										loc->y++;
										if (loc->y >= size)
											{
												loc->y = 0;
												loc->x++;
												if (loc->x >= size)
													{
														pop = 1;
													}
											}
									}
							}
						else
							{
								loc->nx++;
								if (loc->nx >= size)
									{
										loc->nx = 0;
										loc->x++;
										if (loc->x >= size)
											{
												loc->x = 0;
												loc->y++;
												if (loc->y >= size)
													{
														pop = 1;
													}
											}
									}
							}
					}
			return pop;
	}

__device__ int pow2(int x)
	{
		int sum = 1;
		if (x == 0)
			{
				sum = 1;
			}
		else
			{
				for (int i = 0; i < x; i++)
					{
						sum = sum * 2;
					}
			}
		return sum;
	}
__device__ void add(grid ** base, grid ** last, grid * newList)
	{
		if (newList != NULL)
			{
				if (*base == NULL)
					{
						*base = newList;
						*last = *base;
					}
				else
					{
						(*last)->next = newList;
					}
				while ((*last)->next != NULL)
					{
						*last = (*last)->next;
					}
			}
	}

char convert(int x)
	{
		char res = 'a';
		if (x >= 0)
			{
				//int amount = log2((double) x) + (int) res;
				int amount = int(x) + (int) res;

				res = (char) amount;
			}
		else
			{
				res = ' ';
			}
		return res;
	}

__device__ void cloneToGrid(grid * g, grid * g2)
	{
		g2->size = g->size;
		g2->ok = g->ok;
		for (int row = 0; row < g->size; row++)
			{
				for (int col = 0; col < g->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
	}

grid * allocateGrid(int size)
	{
		grid * g2 = NULL;
		allocated += (int) sizeof(grid);
		hipMallocManaged((void **) &g2, sizeof(grid));
		g2->size = size;
		cell ** cells;
		hipMallocManaged((void **) &cells, size * sizeof(cell *));
		allocated += (int) size * sizeof(cell *);
		for (int i = 0; i < size; i++)
			{
				hipMallocManaged((void **) &cells[i], size * sizeof(cell));
				allocated += (int) size * sizeof(cell);
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}

__device__ grid * allocateGridDevice(int size)
	{
		grid * g2 = NULL;
		g2 = (grid *) malloc(sizeof(grid));
		g2->size = size;
		cell ** cells;
		cells = (cell **) malloc(size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				cells[i] = (cell*) malloc(size * sizeof(cell));
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}

__device__ grid * cloneGrid(grid * g)
	{
		grid * g2 = (grid *) malloc(sizeof(grid));
		if (g2 != NULL)
			{
				g2->size = g->size;
				cell * array = (cell *) malloc(g->size * g2->size * sizeof(cell));
				cell ** cells = (cell **) malloc(g2->size * sizeof(cell *));
				for (int i = 0; i < g->size; i++)
					{
						cells[i] = &array[i * g2->size];
					}
				g2->cells = cells;
				for (int row = 0; row < g2->size; row++)
					{
						for (int col = 0; col < g2->size; col++)
							{
								g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
								g2->cells[row][col].value = g->cells[row][col].value;
							}
					}
				g2->next = NULL;
				g2->ok = '0';
			}
		return g2;
	}

__device__ void eliminateValue(cell **c, int row, int col, int max, int value)
	{
		int mask = pow2(value);
		for (int r1 = 0; r1 < max; r1++)
			{
				if (r1 != row)
					{
						c[r1][col].bitmap |= mask;
					}
			}
		for (int c1 = 0; c1 < max; c1++)
			{
				if (c1 != col)
					{
						c[row][c1].bitmap |= mask;
					}
			}
	}

__device__ int check(grid * g, int row, int col, int number)
	{
		int result;
		cell * c = &g->cells[row][col];
		if (c->value >= 0 && c->value != number)
			{
				result = 1;
			}
		else
			{
				int mask = pow2(number);
				int bits = c->bitmap;
				result = (mask & bits);
			}
		return result;
	}

void printGrid(grid * g)
	{
		int n = g->size;
		//printf("X=%d Y=%d %c\n", x, y, g->ok);
		printf("-- Grid -- \n");
		for (int row = 0; row < n; row++)
			{
				printf("%01d# ", row);
				for (int col = 0; col < n; col++)
					{
						cell c = g->cells[row][col];
						//printf("[%02d][%02d]%02X ",row, col, c[row * N + col].bitmap);
						int value = c.value;
						char printC = ' ';
						if (value < 0)
							{
								value = c.bitmap;
								printf(" %03X%c", value, printC);
							}
						else
							{
								//
								printC = convert(value);
								value = 0;
								printf("  %c  ", printC);
							}

					}
				printf("\n");
			}
	}
