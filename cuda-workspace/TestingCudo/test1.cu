
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#define N 10
#define UP 'U'
#define LEFT 'L'
#define MAX 4

typedef struct path
	{
	struct path * next;
	char direction;
	int letters[4];
	char * domain;
	char * pass;
	} path;

struct cell
	{
	int value;
	int bitmap;
	};

typedef struct grid
	{
	cell ** cells;
	//cell cells[N][N];
	short size;
	struct grid * next;
	//int xx;
	char ok;
	//int yy;
	} grid;
typedef struct location
{
	int x;
	int nx;
	int y;
	int ny;
	struct location * next;
	struct grid * currentG;
	path * p;
}location;

void initCell(cell * c);
__device__ int pow2(int x);
__device__ void printGrid(grid * g);
__device__ char convert(int x);
__global__ void compute(grid * g, path * p, location * loc);
__device__ void computeIterative(grid * g, path * p, location * loc);
__device__ void cloneToGrid(grid * g, grid * g2);
__device__ void eliminateValue(cell **c, int row, int col, int max, int value);
__device__ void add(grid ** base, grid ** last, grid * newList);
__device__ int check(grid * g, int row, int col, int number);
__device__	grid * allocateGridDevice(int size);
//void printGrid(grid * g, int x, int y);
__device__ grid * cloneGrid(grid * g);
grid * allocateGrid(int size);
path * getPath(char * line);
void printPath(path * p);
path ** scanChars();
int foo(path * p);

int main(void)
	{
		path ** p = scanChars();
		if (p != NULL)
			{
				foo(p[1]);
			}
	}

__global__ void compute(grid * g, path * p,location * l)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N * N)
			{
				//int x = blockIdx.x;
				//int y = threadIdx.x;
				computeIterative(g, p, l);
			}
	}
__device__ void computeIterative(grid * g, path * p, location * loc)
	{
		int i = 0;
		if (p->direction == LEFT) //Do UP/DOWN
		{
			loc->nx = 0;
			loc->ny = loc->y;
		}						
		else
		{
			loc->nx = loc->x;
			loc->ny = 0;
		}
		location * temp; 						
		int checkValue;
		int value;
		int z;
		int done = 0;
		//int idx = blockIdx.x * blockDim.x + threadIdx.x;
		//int base = idx * MAX *3 + recCount;
		grid * currentGrid = allocateGridDevice(g->size);
		cloneToGrid(g,currentGrid);
		loc->currentG = allocateGridDevice(g->size);
		loc->p = p;
		//recCount = recCount +3 ;
		//int set = 0;
		int x,y;
		int count = 0;
		while(done == 0)
		{
			cloneToGrid(loc->currentG, currentGrid);
			x = loc->x;
			y = loc->y;
			
			p = loc->p;
			int lasty = y;
			int lastx = x;
			value = p->letters[0];
			checkValue = check(currentGrid, x, y, value);
			if (checkValue == 0)
			{
				currentGrid->cells[x][y].value = value;
				eliminateValue(currentGrid->cells, x, y, currentGrid->size, value);
				int direction;
				if (p->direction == LEFT) //Do UP/DOWN
				{
					z = loc->ny;
					loc->ny++;
				}
				else
				{
					z = loc->nx;
					loc->nx++;
				}
				checkValue = 0;
				if (p->direction == LEFT) //Do UP/DOWN
					direction = lasty > z ? -1 : 1;
				else
					direction = lastx > z ? -1 : 1;
				for (int offset = 0; offset < 3 && checkValue == 0; offset++)
					{
						value = p->letters[offset + 1];
						if (p->direction == LEFT) //Do UP/DOWN
							lasty = (z + (offset * direction) + currentGrid->size) % currentGrid->size;
						else
							lastx = (z + (offset * direction) + currentGrid->size) % currentGrid->size;
						checkValue |= check(currentGrid, lastx, lasty, value);
						if (checkValue == 0)
							{
								currentGrid->cells[lastx][lasty].value = value;
								eliminateValue(currentGrid->cells, lastx, lasty, currentGrid->size, value);
							}
					}
					//if(checkValue == 0)
					//{
					//	printGrid(currentGrid);	
					//}
					
				}
			if(checkValue == 0 && count == MAX)
			{
				i ++;
				//printGrid(currentGrid);
			}
			if (checkValue == 0 && count < MAX) //rec value
				{
				
							//cloneToGrid(currentGrid, res[base]
				
				if(p->next != NULL)
					{
						temp = (location *)malloc(sizeof(location));
						temp->x = lastx;
						temp->y = lasty;
						if (p->next->direction == LEFT) //Do UP/DOWN
						{
							temp->nx = 0;
							temp->ny = temp->y;
						}						
						else
						{
							temp->nx = temp->x;
							temp->ny = 0;
						}
						//printf("Next x=%d y=%d nx=%d ny=%d\n",temp->x,temp->y,temp->nx,temp->ny);
						temp->p = p->next;
						temp->currentG = allocateGridDevice(g->size);
						cloneToGrid(currentGrid,temp->currentG);
						temp->next = loc;
						loc = temp;
						
						count ++;
					}
				}
			else
			{
			if(p->direction == LEFT)
			{
				z = loc->ny;
			}
			else
			{
				z = loc->nx;
			}
			if(z == g->size)
				{
					if(loc->next == NULL)
					{
						done = 1;
					}
					else
					{
						temp = loc->next;
						free(loc->currentG);
						free(loc);
						loc = temp;
						count --;
					}
			}
			}	
		}
		printf("The total is %d\n",i);
	}



__device__ void eliminateValue(cell **c, int row, int col, int max, int value)
	{
		int mask = pow2(value);
		for (int r1 = 0; r1 < max; r1++)
			{
				if (r1 != row)
					{
						c[r1][col].bitmap |= mask;
					}
			}
		for (int c1 = 0; c1 < max; c1++)
			{
				if (c1 != col)
					{
						c[row][c1].bitmap |= mask;
					}
			}
	}

__device__ int check(grid * g, int row, int col, int number)
	{
		int result;
		cell * c = &g->cells[row][col];
		if (c->value >= 0 && c->value != number)
			{
				result = 1;
			}
		else
			{
				int mask = pow2(number);
				int bits = c->bitmap;
				result = (mask & bits);
			}
		return result;
	}

int foo(path * p)
	{
		hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024*8); //See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#heap-memory-allocation
		int nBYn = N * N;
		int size = N;
		grid * g = allocateGrid(size);

		int i = 0;
		location * larray;
		hipMallocManaged((void **) &larray,sizeof(location) * nBYn);
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{
						int offset = row * N + col;
						larray[offset].x = row;
						larray[offset].y = col;
					}
			}

		printPath(p);
		compute<<<1, 1>>>(g, p, larray);
		hipDeviceSynchronize();
		/*i = 0;
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{

						for(int j = 0; j <MAX * 3; j+=3)
						{
						int idx = (row * size + col) * MAX*3 +j;
						if (result[idx]->ok == '1')
							{
								printf("(%d,%d,%d)\n", row, col, j);
								printGrid(result[idx]);
							}
						i++;
						}
					}
			}
			*/
		/*for( int i = 0; i < nBYn * MAX * 3; i++)
		{
			if (result[i]->ok == '1')
							{
								//printf("(%d,%d,%d)\n", row, col, j);
								puts("");
								printGrid(result[i]);
							}
		}
*/
		//cudaFree(array);
		return 0;
	}

__device__ void add(grid ** base, grid ** last, grid * newList)
	{
		if (newList != NULL)
			{
				if (*base == NULL)
					{
						*base = newList;
						*last = *base;
					}
				else
					{
						(*last)->next = newList;
					}
				while ((*last)->next != NULL)
					{
						*last = (*last)->next;
					}
			}
	}

/*
 __global__ void add( int *a, int *b, int *c )
 {
 int tid = blockIdx.x; // handle the data at this index
 if (tid < N)
 c[tid] = a[tid] + b[tid];
 }*/

int convertUpper(char u)
	{
		int x = (int) u;
		int A = (int) 'A';
		x = x - A;
		return x;
	}
char convertChar(char u)
	{
		int x = (int) u;
		int A = (int) 'A';
		x = x + A;
		return (char) x;
	}

__device__ char convert(int x)
	{
		char res = 'a';
		if (x >= 0)
			{
				//int amount = log2((double) x) + (int) res;
				int amount = int(x) + (int) res;

				res = (char) amount;
			}
		else
			{
				res = ' ';
			}
		return res;
	}

__device__ void cloneToGrid(grid * g, grid * g2)
	{
		g2->size = g->size;
		g2->ok = g->ok;
		for (int row = 0; row < g->size; row++)
			{
				for (int col = 0; col < g->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
	}


path * allocate(char c, char c1, char* c2, int direction)
	{
		path *p;
		hipMallocManaged((void **) &p, 1);
		p->next = NULL;
		p->letters[0] = convertUpper(c);
		p->letters[1] = convertUpper(c1);
		p->letters[2] = convertUpper(c2[0]);
		p->letters[3] = convertUpper(c2[1]);
		p->direction = direction;
		p->domain = NULL;
		p->pass = NULL;
		return p;
	}
void printPath(path * p)
	{
		if (p != NULL)
			{
				char dir = p->direction == UP ? 'U' : 'L';
				if (p->domain != NULL)
					{
						printf("[%s]->[%s]\n", p->domain, p->pass);
					}
				int value = (int) p->letters[0];
				if (value > 30)
					{
						printf("[%c]->[%c%c%c]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);
					}
				else
					{
						printf("[%c]->[%c%c%c]%c\n", convertChar(p->letters[0]), convertChar(p->letters[1]), convertChar(p->letters[2]), convertChar(p->letters[3]), dir);
						//printf("[%d]->[%d%d%d]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);

					}
				printPath(p->next);
			}
	}
path * getPath(char * line)
	{
		char * domain = line;
		char * pass = strstr(domain, "-");
		long offset = (long) pass - (long) domain;
		domain[offset - 2] = 0;
		pass += 3;

		int domainLen = strlen(domain);

		//printf("[%s]->[%s]\n", domain, pass);
		path * head = NULL;
		path * tail = NULL;
		char previous = domain[domainLen - 1];
		int direction = domainLen % 2 == 1 ? UP : LEFT;
		for (int i = 0; i < domainLen; i++)
			{
				char next = domain[i];
				path * current = allocate(previous, next, &pass[i * 2], direction);
				previous = pass[i*2+1];
				if (head == NULL)
					{
						head = current;
						hipMallocManaged((void **) &head->domain, domainLen + 1);
						strcpy(head->domain, domain);
						hipMallocManaged((void **) &head->pass, strlen(pass) + 1);
						strcpy(head->pass, pass);
					}
				if (tail == NULL)
					{
						tail = head;
					}
				else
					{
						tail->next = current;
						tail = current;
					}
				direction = direction == UP ? LEFT : UP;
			}

		return head;
	}
path ** scanChars()
	{
		char test[100] = "HEBJCE  -> BJAGDHCHJEGJ";
		int count = 100;
		int index = 0;
		path ** pathList;
		hipMallocManaged((void **) &pathList, (sizeof(path *)) * count);
		path * p = getPath(test);
		printPath(p);
		FILE * database;
		char buffer[30];

		database = fopen("output.txt", "r");

		if (NULL == database)
			{
				perror("opening database");
				return NULL;
			}
		int max = 10;
		while (EOF != fscanf(database, "%[^\n]\n", buffer) && index < max)
			{
				char * b = buffer;
				while (*b != '-')
					{
						if (*b == 0)
							{
								*b = ' ';
							}
						b++;
					}
				if (index < max)
					{
						p = getPath(buffer);
						pathList[index] = p;
						index++;
						if (index == count)
							{
								path ** temp;
								hipMallocManaged((void **) &temp, (sizeof(path *) * count * 1.5));
								for (int i = 0; i < count - 1; i++)
									{
										temp[i] = pathList[i];
									}
								hipFree(pathList);
								pathList = temp;
								count *= 1.5;

							}

						//printf("> %s\n", buffer);
						//	getPath(buffer);
					}
			}

		/*for (int i = 0; i < count; i++)
		 {
		 printPath(pathList[i]);
		 }*/
		printf("Count is  = %d\n", index);
		fclose(database);
		return pathList;
	}





__device__ int pow2(int x)
{
	int sum = 1;
	if( x == 0)
	{
		sum = 1;
	}
	else
	{
	for(int i = 0; i < x; i++)
	{
		sum = sum *2;
	}
	}
	return sum;
}

__device__ void printGrid(grid * g)
	{
		int n = g->size;
		//printf("X=%d Y=%d %c\n", x, y, g->ok);
		for (int row = 0; row < n; row++)
			{
				for (int col = 0; col < n; col++)
					{
						cell c = g->cells[row][col];
						//printf("[%02d][%02d]%02X ",row, col, c[row * N + col].bitmap);
						int value = c.value;
						char printC = ' ';
						if (value < 0)
							{
								value = c.bitmap;
								printf(" %03X%c", value, printC);
							}
						else
							{
								//
								printC = convert(value);
								value = 0;
								printf("  %c  ", printC);
							}

					}
				printf("\n");
			}
	}

	grid * allocateGrid(int size)
	{
		grid * g2 = NULL;
		hipMallocManaged((void **) &g2, sizeof(grid));
		g2->size = size;
		cell ** cells;
		hipMallocManaged((void **) &cells, size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				hipMallocManaged((void **) &cells[i], size * sizeof(cell));
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}
__device__	grid * allocateGridDevice(int size)
	{
		grid * g2 = NULL;
		g2 = (grid *) malloc(sizeof(grid));
		g2->size = size;
		cell ** cells;
		cells = (cell **) malloc(size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				cells[i] = (cell*) malloc(size * sizeof(cell));
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}
__device__ grid * cloneGrid(grid * g)
	{
		grid * g2 = (grid *) malloc(sizeof(grid));
		if(g2 != NULL)
		{
		g2->size = g->size;
		cell * array = (cell *) malloc(g->size * g2->size * sizeof(cell));
		cell ** cells = (cell **) malloc(g2->size * sizeof(cell *));
		for (int i = 0; i < g->size; i++)
			{
				cells[i] = &array[i * g2->size];
			}
		g2->cells = cells;
		for (int row = 0; row < g2->size; row++)
			{
				for (int col = 0; col < g2->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		}
		return g2;
	}
