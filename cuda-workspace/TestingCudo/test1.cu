#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include "grid.h"
#define MAX 1
#define N 10
void initCell(cell * c);
__global__ void compute2(grid * g, path * p, location * loc);
__device__ void computeIterative(grid ** result, int gridSize,grid * g, path * p, location * loc);
__device__ void add(grid ** base, grid ** last, grid * newList);
__device__ void cloneToGrid(grid * g, grid * g2);
__device__ void eliminateValue(cell **c, int row, int col, int max, int value);
__device__ int check(grid * g, int row, int col, int number);
__device__	grid * allocateGridDevice(int size);
__device__ void printGrid(grid * g);
__device__ int pow2(int x);
__device__ grid * cloneGrid(grid * g);
__device__ char convert(int x);
int foo(path * p);
int main(void)
	{
		int deviceCount;
		hipGetDeviceCount(&deviceCount);
		int device;
		for (device = 0; device < deviceCount; ++device)
		{
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, device);
			printf("Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor);
		}//	 - See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/#multi-device-system
		//hipSetDevice(1);
		path ** p = scanChars();
		if (p != NULL)
			{
				foo(p[1]);
			}
	}
__global__ void compute2(grid ** result, int gridSize, grid * g, path * p,location * l)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N * N)
			{
				//int x = blockIdx.x;
				//int y = threadIdx.x;
				computeIterative(result, gridSize, g, p, l);
			}
	}
__device__ void computeIterative(grid ** result, int gridSize,grid * g, path * p, location * loc)
	{
		int breaker = 0;
		int bmax = 2;
		int printcount = 0;
		location * freeHead = NULL;
		location * freeTail = NULL;
		int i = 0;
		if (p->direction == LEFT) //Do UP/DOWN
		{
			loc->nx = 0;
			loc->ny = loc->y;
		}
		else
		{
			loc->nx = loc->x;
			loc->ny = 0;
		}
		location * temp;
		int checkValue;
		int value;
		int z;
		int done = 0;
		//int idx = blockIdx.x * blockDim.x + threadIdx.x;
		//int base = idx * MAX *3 + recCount;
		grid * currentGrid = allocateGridDevice(g->size);
		cloneToGrid(g,currentGrid);
		loc->currentG = allocateGridDevice(g->size);
		loc->p = p;
		//recCount = recCount +3 ;
		//int set = 0;
		int x,y;
		int count = 0;
		while(done == 0)
		{
			breaker++;
			cloneToGrid(loc->currentG, currentGrid);
			x = loc->x;
			y = loc->y;

			p = loc->p;
			int lasty = y;
			int lastx = x;
			value = p->letters[0];
			checkValue = check(currentGrid, x, y, value);
			if (checkValue == 0)
			{
				currentGrid->cells[x][y].value = value;
				eliminateValue(currentGrid->cells, x, y, currentGrid->size, value);
				int direction;
				if (p->direction == LEFT) //Do UP/DOWN
				{
					z = loc->ny;
					loc->ny++;
				}
				else
				{
					z = loc->nx;
					loc->nx++;
				}
				checkValue = 0;
				if (p->direction == LEFT) //Do UP/DOWN
					direction = lasty > z ? -1 : 1;
				else
					direction = lastx > z ? -1 : 1;
				for (int offset = 0; offset < 3 && checkValue == 0; offset++)
					{
						value = p->letters[offset + 1];
						if (p->direction == LEFT) //Do UP/DOWN
							lasty = (z + (offset * direction) + currentGrid->size) % currentGrid->size;
						else
							lastx = (z + (offset * direction) + currentGrid->size) % currentGrid->size;
						checkValue |= check(currentGrid, lastx, lasty, value);
						if (checkValue == 0)
							{
								currentGrid->cells[lastx][lasty].value = value;
								eliminateValue(currentGrid->cells, lastx, lasty, currentGrid->size, value);
							}
					}
					//if(checkValue == 0)
					//{
					//	printGrid(currentGrid);
					//}

				}
			if(checkValue == 0 && count == MAX)
			{
				i ++;
				if(printcount < gridSize)
				{
					cloneToGrid(currentGrid,result[printcount]);
					result[printcount].ok = '1';
					//printGrid(currentGrid);
					printcount++;
				}
				else
				{
					done =1 ;
				}
			}
			if (checkValue == 0) //rec value
				{
							//cloneToGrid(currentGrid, res[base]
				if(p->next != NULL && count < MAX)
					{
						if(freeHead == NULL)
						{
							temp = (location *)malloc(sizeof(location));
							temp->currentG = allocateGridDevice(g->size);
							printf("Allocated Block\n");
						}
						else
						{
							temp = freeHead;
							if(freeHead->next != NULL)
							{
								freeHead = freeHead->next;
							}
						}
						temp->x = lastx;
						temp->y = lasty;
						if (p->next->direction == LEFT) //Do UP/DOWN
						{
							temp->nx = 0;
							temp->ny = temp->y;
						}
						else
						{
							temp->nx = temp->x;
							temp->ny = 0;
						}
						//printf("Next x=%d y=%d nx=%d ny=%d\n",temp->x,temp->y,temp->nx,temp->ny);
						temp->p = p->next;

						cloneToGrid(currentGrid,temp->currentG);
						temp->next = loc;
						loc = temp;

						count ++;
					}
				}
			else
			{
			if(p->direction == LEFT)
			{
				z = loc->ny;
			}
			else
			{
				z = loc->nx;
			}
			if(z == g->size)
				{
					if(loc->next == NULL)
					{
						done = 1;
					}
					else
					{
						temp = loc->next;
						if(freeHead == NULL)
						{
							freeHead = loc;
							freeTail = loc;
						}
						else
						{
							freeTail->next = loc;
							freeTail = loc;
							freeTail->next = NULL;
						}
						//free(loc->currentG);
						//free(loc);
						loc = temp;
						count --;
					}
			}
			}
		if(bmax == breaker)
		{
			printf("Breaker %d\n",breaker);
			bmax *= 2;
		}
		if(breaker == 2147483648)
		{
			done = 1;
			printf("Breaker Max hit!");
		}
		}
		printf("The total is %d\n",i);

	}



int foo(path * p)
	{
		hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024*8); //See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#heap-memory-allocation
		int nBYn = N * N;
		int size = N;
		grid * g = allocateGrid(size);
		int gridSize = 10000;
		grid ** result;
		int i;
		hipMallocManaged((void **) &larray,sizeof(grid *) * gridSize);
		for(i = 0; i < gridSize; i++)
		{
			result[i] = allocateGrid(size);
		}
		//int i = 0;
		location * larray;
		hipMallocManaged((void **) &larray,sizeof(location) * nBYn);
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{
						int offset = row * N + col;
						larray[offset].x = row;
						larray[offset].y = col;
					}
			}

		printPath(p);
		compute2<<<1, 1>>>(result,gridSize, g, p, larray);
		hipDeviceSynchronize();
		for(i = 0; i < gridSize; i++)
		{
			if(result[i].ok == '1')
			{
				printGrid(result[i]);
			}
		}
		/*i = 0;
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{

						for(int j = 0; j <MAX * 3; j+=3)
						{
						int idx = (row * size + col) * MAX*3 +j;
						if (result[idx]->ok == '1')
							{
								printf("(%d,%d,%d)\n", row, col, j);
								printGrid(result[idx]);
							}
						i++;
						}
					}
			}
		*/	
		/*for( int i = 0; i < nBYn * MAX * 3; i++)
		{
			if (result[i]->ok == '1')
							{
								//printf("(%d,%d,%d)\n", row, col, j);
								puts("");
								printGrid(result[i]);
							}
		}
*/
		//hipFree(array);
		return 0;
	}


__device__ int pow2(int x)
{
	int sum = 1;
	if( x == 0)
	{
		sum = 1;
	}
	else
	{
	for(int i = 0; i < x; i++)
	{
		sum = sum *2;
	}
	}
	return sum;
}
__device__ void add(grid ** base, grid ** last, grid * newList)
	{
		if (newList != NULL)
			{
				if (*base == NULL)
					{
						*base = newList;
						*last = *base;
					}
				else
					{
						(*last)->next = newList;
					}
				while ((*last)->next != NULL)
					{
						*last = (*last)->next;
					}
			}
	}

__device__ char convert(int x)
	{
		char res = 'a';
		if (x >= 0)
			{
				//int amount = log2((double) x) + (int) res;
				int amount = int(x) + (int) res;

				res = (char) amount;
			}
		else
			{
				res = ' ';
			}
		return res;
	}

__device__ void cloneToGrid(grid * g, grid * g2)
	{
		g2->size = g->size;
		g2->ok = g->ok;
		for (int row = 0; row < g->size; row++)
			{
				for (int col = 0; col < g->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
	}


	grid * allocateGrid(int size)
	{
		grid * g2 = NULL;
		hipMallocManaged((void **) &g2, sizeof(grid));
		g2->size = size;
		cell ** cells;
		hipMallocManaged((void **) &cells, size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				hipMallocManaged((void **) &cells[i], size * sizeof(cell));
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}

__device__	grid * allocateGridDevice(int size)
	{
		grid * g2 = NULL;
		g2 = (grid *) malloc(sizeof(grid));
		g2->size = size;
		cell ** cells;
		cells = (cell **) malloc(size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				cells[i] = (cell*) malloc(size * sizeof(cell));
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}

__device__ grid * cloneGrid(grid * g)
	{
		grid * g2 = (grid *) malloc(sizeof(grid));
		if(g2 != NULL)
		{
		g2->size = g->size;
		cell * array = (cell *) malloc(g->size * g2->size * sizeof(cell));
		cell ** cells = (cell **) malloc(g2->size * sizeof(cell *));
		for (int i = 0; i < g->size; i++)
			{
				cells[i] = &array[i * g2->size];
			}
		g2->cells = cells;
		for (int row = 0; row < g2->size; row++)
			{
				for (int col = 0; col < g2->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		}
		return g2;
	}

__device__ void eliminateValue(cell **c, int row, int col, int max, int value)
	{
		int mask = pow2(value);
		for (int r1 = 0; r1 < max; r1++)
			{
				if (r1 != row)
					{
						c[r1][col].bitmap |= mask;
					}
			}
		for (int c1 = 0; c1 < max; c1++)
			{
				if (c1 != col)
					{
						c[row][c1].bitmap |= mask;
					}
			}
	}

__device__ int check(grid * g, int row, int col, int number)
	{
		int result;
		cell * c = &g->cells[row][col];
		if (c->value >= 0 && c->value != number)
			{
				result = 1;
			}
		else
			{
				int mask = pow2(number);
				int bits = c->bitmap;
				result = (mask & bits);
			}
		return result;
	}

__device__ void printGrid(grid * g)
	{
		int n = g->size;
		//printf("X=%d Y=%d %c\n", x, y, g->ok);
		for (int row = 0; row < n; row++)
			{
				for (int col = 0; col < n; col++)
					{
						cell c = g->cells[row][col];
						//printf("[%02d][%02d]%02X ",row, col, c[row * N + col].bitmap);
						int value = c.value;
						char printC = ' ';
						if (value < 0)
							{
								value = c.bitmap;
								printf(" %03X%c", value, printC);
							}
						else
							{
								//
								printC = convert(value);
								value = 0;
								printf("  %c  ", printC);
							}

					}
				printf("\n");
			}
	}
