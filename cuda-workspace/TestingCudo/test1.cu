#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include "grid.h"
#define N 10
#define UP 'U'
#define LEFT 'L'
#define MAX 3
void initCell(cell * c);
__global__ void compute2(grid * g, path * p, location * loc);
__device__ void computeIterative(grid * g, path * p, location * loc);
int foo(path * p);
int main(void)
	{
		int deviceCount;
		hipGetDeviceCount(&deviceCount);
		int device;
		for (device = 0; device < deviceCount; ++device)
		{
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, device);
			printf("Device %d has compute capability %d.%d.\n", device, deviceProp.major, deviceProp.minor);
		}//	 - See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/#multi-device-system
		//hipSetDevice(1);
		path ** p = scanChars();
		if (p != NULL)
			{
				foo(p[1]);
			}
	}

__global__ void compute2(grid * g, path * p,location * l)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N * N)
			{
				//int x = blockIdx.x;
				//int y = threadIdx.x;
				computeIterative(g, p, l);
			}
	}
__device__ void computeIterative(grid * g, path * p, location * loc)
	{
		int breaker = 0;
		int bmax = 2;
		location * freeHead = NULL;
		location * freeTail = NULL;
		int i = 0;
		if (p->direction == LEFT) //Do UP/DOWN
		{
			loc->nx = 0;
			loc->ny = loc->y;
		}
		else
		{
			loc->nx = loc->x;
			loc->ny = 0;
		}
		location * temp;
		int checkValue;
		int value;
		int z;
		int done = 0;
		//int idx = blockIdx.x * blockDim.x + threadIdx.x;
		//int base = idx * MAX *3 + recCount;
		grid * currentGrid = allocateGridDevice(g->size);
		cloneToGrid(g,currentGrid);
		loc->currentG = allocateGridDevice(g->size);
		loc->p = p;
		//recCount = recCount +3 ;
		//int set = 0;
		int x,y;
		int count = 0;
		while(done == 0)
		{
			breaker++;
			cloneToGrid(loc->currentG, currentGrid);
			x = loc->x;
			y = loc->y;

			p = loc->p;
			int lasty = y;
			int lastx = x;
			value = p->letters[0];
			checkValue = check(currentGrid, x, y, value);
			if (checkValue == 0)
			{
				currentGrid->cells[x][y].value = value;
				eliminateValue(currentGrid->cells, x, y, currentGrid->size, value);
				int direction;
				if (p->direction == LEFT) //Do UP/DOWN
				{
					z = loc->ny;
					loc->ny++;
				}
				else
				{
					z = loc->nx;
					loc->nx++;
				}
				checkValue = 0;
				if (p->direction == LEFT) //Do UP/DOWN
					direction = lasty > z ? -1 : 1;
				else
					direction = lastx > z ? -1 : 1;
				for (int offset = 0; offset < 3 && checkValue == 0; offset++)
					{
						value = p->letters[offset + 1];
						if (p->direction == LEFT) //Do UP/DOWN
							lasty = (z + (offset * direction) + currentGrid->size) % currentGrid->size;
						else
							lastx = (z + (offset * direction) + currentGrid->size) % currentGrid->size;
						checkValue |= check(currentGrid, lastx, lasty, value);
						if (checkValue == 0)
							{
								currentGrid->cells[lastx][lasty].value = value;
								eliminateValue(currentGrid->cells, lastx, lasty, currentGrid->size, value);
							}
					}
					//if(checkValue == 0)
					//{
					//	printGrid(currentGrid);
					//}

				}
			if(checkValue == 0 && count == MAX)
			{
				i ++;
				//printGrid(currentGrid);
			}
			if (checkValue == 0 && count < MAX) //rec value
				{

							//cloneToGrid(currentGrid, res[base]

				if(p->next != NULL)
					{
						if(freeHead == NULL)
						{
							temp = (location *)malloc(sizeof(location));
							temp->currentG = allocateGridDevice(g->size);
							printf("Allocated Block\n");
						}
						else
						{
							temp = freeHead;
							if(freeHead->next != NULL)
							{
								freeHead = freeHead->next;
							}
						}
						temp->x = lastx;
						temp->y = lasty;
						if (p->next->direction == LEFT) //Do UP/DOWN
						{
							temp->nx = 0;
							temp->ny = temp->y;
						}
						else
						{
							temp->nx = temp->x;
							temp->ny = 0;
						}
						//printf("Next x=%d y=%d nx=%d ny=%d\n",temp->x,temp->y,temp->nx,temp->ny);
						temp->p = p->next;
						
						cloneToGrid(currentGrid,temp->currentG);
						temp->next = loc;
						loc = temp;

						count ++;
					}
				}
			else
			{
			if(p->direction == LEFT)
			{
				z = loc->ny;
			}
			else
			{
				z = loc->nx;
			}
			if(z == g->size)
				{
					if(loc->next == NULL)
					{
						done = 1;
					}
					else
					{
						temp = loc->next;
						if(freeHead == NULL)
						{
							freeHead = loc;
							freeTail = loc;
						}
						else
						{
							freeTail->next = loc;
							freeTail = loc;
							freeTail->next = NULL;
						}
						//free(loc->currentG);
						//free(loc);
						loc = temp;
						count --;
					}
			}
			}
		if(bmax == breaker)
		{
			printf("Breaker %d\n",breaker);
			bmax *= 2;
		}
		if(breaker == 2147483648)
		{
			done = 1;
			printf("Breaker Max hit!");
		}
		}
		printf("The total is %d\n",i);
		
	}



int foo(path * p)
	{
		hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024*8); //See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#heap-memory-allocation
		int nBYn = N * N;
		int size = N;
		grid * g = allocateGrid(size);

		//int i = 0;
		location * larray;
		hipMallocManaged((void **) &larray,sizeof(location) * nBYn);
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{
						int offset = row * N + col;
						larray[offset].x = row;
						larray[offset].y = col;
					}
			}

		printPath(p);
		compute2<<<1, 1>>>(g, p, larray);
		hipDeviceSynchronize();
		/*i = 0;
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{

						for(int j = 0; j <MAX * 3; j+=3)
						{
						int idx = (row * size + col) * MAX*3 +j;
						if (result[idx]->ok == '1')
							{
								printf("(%d,%d,%d)\n", row, col, j);
								printGrid(result[idx]);
							}
						i++;
						}
					}
			}
			*/
		/*for( int i = 0; i < nBYn * MAX * 3; i++)
		{
			if (result[i]->ok == '1')
							{
								//printf("(%d,%d,%d)\n", row, col, j);
								puts("");
								printGrid(result[i]);
							}
		}
*/
		//hipFree(array);
		return 0;
	}



