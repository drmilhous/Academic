
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 10
#define UP 'U'
#define LEFT 'L'
#define MAX 8

typedef struct path
	{
	struct path * next;
	char direction;
	int letters[4];
	char * domain;
	char * pass;
	} path;

struct cell
	{
	int value;
	int bitmap;
	};

typedef struct grid
	{
	cell ** cells;
	//cell cells[N][N];
	short size;
	struct grid * next;
	//int xx;
	char ok;
	//int yy;
	} grid;
typedef struct state
{
	int idx;
	grid * currentGrid;
	int base;
	int lastx;
	int lasty;
	int x1;
	int y1;
	int direction;
	int checkValue;
	int set;
	int value;
	int offset;
	int row;
	int col;
}state;
void initCell(cell * c);
char convert(int x);
__device__ void computeRecursive(grid * g, path * p,path** p2, int x, int y, grid ** res, int recCount);
__device__ void cloneToGrid(grid * g, grid * g2);
__device__ void eliminateValue(cell **c, int row, int col, int max, int value);
__device__ void add(grid ** base, grid ** last, grid * newList);
//void printGrid(grid * g, int x, int y);
__device__ grid * cloneGrid(grid * g);
void printGrid(grid * g)
	{
		int n = g->size;
		//printf("X=%d Y=%d %c\n", x, y, g->ok);
		for (int row = 0; row < n; row++)
			{
				for (int col = 0; col < n; col++)
					{
						cell c = g->cells[row][col];
						//printf("[%02d][%02d]%02X ",row, col, c[row * N + col].bitmap);
						int value = c.value;
						char printC = ' ';
						if (value < 0)
							{
								value = c.bitmap;
								printf(" %03X%c", value, printC);
							}
						else
							{
								//
								printC = convert(value);
								value = 0;
								printf("  %c  ", printC);
							}
						
					}
				printf("\n");
			}
	}
	__device__ int pow2(int x)
{
	int sum = 1;
	if( x == 0)
	{
		sum = 1;
	}
	else
	{
	for(int i = 0; i < x; i++)
	{
		sum = sum *2;
	}
	}
	return sum;
}
__device__ void eliminateValue(cell **c, int row, int col, int max, int value)
	{
		//int mask = pow(2.0, (double) value);
		int mask = pow2(value);
		for (int r1 = 0; r1 < max; r1++)
			{
				if (r1 != row)
					{
						c[r1][col].bitmap |= mask;
					}
			}
		for (int c1 = 0; c1 < max; c1++)
			{
				if (c1 != col)
					{
						c[row][c1].bitmap |= mask;
					}
				//x->c = ch;
			}
	}

__device__ int check(grid * g, int row, int col, int number)
	{
		int result;
		cell * c = &g->cells[row][col];
		if (c->value >= 0 && c->value != number)
			{
				result = 1;
			}
		else
			{
				//int mask = pow(2.0, (double) number);
				int mask = pow2(number);
				int bits = c->bitmap;
				result = (mask & bits);
			}
		return result;
	}
grid * allocateGrid(int size)
	{

		grid * g2 = NULL;
		hipMallocManaged((void **) &g2, sizeof(grid));
		g2->size = size;
		//cell * array;
		//cudaMallocManaged((void **) &array, size * size * sizeof(cell));
		cell ** cells;
		hipMallocManaged((void **) &cells, size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				//cells[i] = &array[i * size];
				hipMallocManaged((void **) &cells[i], size * sizeof(cell));
			}
		g2->cells = cells;

		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		//printf("XX%p->%c\n",g2, g2->ok);
		return g2;
	}
__device__ grid * cloneGrid(grid * g)
	{
		grid * g2 = (grid *) malloc(sizeof(grid));
		if(g2 != NULL)
		{
		g2->size = g->size;
		cell * array = (cell *) malloc(g->size * g2->size * sizeof(cell));
		cell ** cells = (cell **) malloc(g2->size * sizeof(cell *));
		for (int i = 0; i < g->size; i++)
			{
				cells[i] = &array[i * g2->size];
			}
		g2->cells = cells;
		for (int row = 0; row < g2->size; row++)
			{
				for (int col = 0; col < g2->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		}
		return g2;
	}
__global__ void compute(grid * g, path * p,path ** p2, grid ** result)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N * N)
			{
				int x = blockIdx.x;
				int y = threadIdx.x;
				computeRecursive(g, p,p2, x, y, result, 0);
			}
	}
__device__ void computeRecursive(grid * g, path * p, path ** nextPath, int x, int y, grid ** res, int recCount)
	{
		state * s = (state *) malloc(sizeof(state));
		s->idx = blockIdx.x * blockDim.x + threadIdx.x;
		s->base = s->idx * MAX *2 + recCount;
		//printf("index[%02d] base[%d]\n",idx, base);
		s->currentGrid = res[s->base +1];
		recCount = recCount +2 ;
		s->set = 0;
		s->checkValue = 0;
		s->value = p->letters[0];
		cloneToGrid(g,s->currentGrid);
		if(s->currentGrid != NULL)
		{
		//grid* previousGrid = res[base + 2];
		s->checkValue = check(s->currentGrid, x, y, s->value);
		if (s->checkValue == 0)
			{
				s->currentGrid->cells[x][y].value = s->value;
				eliminateValue(s->currentGrid->cells, x, y, s->currentGrid->size, s->value);
				//cloneToGrid(currentGrid, previousGrid);
				if (p->direction == LEFT) //Do UP/DOWN
					{
						s->lasty = y;
						for (s->y1 = 0; s->y1 < s->currentGrid->size; (s->y1)++) //check above
							{
								if (s->y1 != y)
									{
										s->direction = y > s->y1 ? -1 : 1;
										s->checkValue = 0;
										for (s->offset = 0; s->offset < 3; (s->offset)++)
											{
												s->value = p->letters[s->offset + 1];
												s->lasty = (s->y1 + (s->offset * s->direction) + s->currentGrid->size) % s->currentGrid->size;
												s->checkValue |= check(s->currentGrid, x, s->lasty, s->value);
												if (s->checkValue == 0)
													{
														s->currentGrid->cells[x][s->lasty].value = s->value;
														eliminateValue(s->currentGrid->cells, x, s->lasty, s->currentGrid->size, s->value);
													}
											}
										if (s->checkValue == 0) //recursive call
											{
												if (s->set == 0)
													{
														s->set = 1;
														cloneToGrid(s->currentGrid, res[s->base]);
														res[s->base]->ok = '1';
													}
												if(recCount < MAX *2)
												 {
													if (p->next != NULL )
													{
														computeRecursive(s->currentGrid, p->next,nextPath, x, s->lasty, res, recCount);
														//add(&result, &last, temp);
													}
													else
													{
														printf("Next Path %d\n", s->idx );
														p = nextPath[0];
														nextPath++;
														for (s->row = 0; s->row < g->size; s->row++)
														{
															for (s->col = 0; s->col < g->size; s->col++)
																{
																	computeRecursive(s->currentGrid, p,nextPath, s->row, s->col, res, recCount);
																}
														}	
													}
												 }
											}
									}
								//cloneToGrid(previousGrid, currentGrid);
								cloneToGrid(g,s->currentGrid);
								s->currentGrid->cells[x][y].value = p->letters[0];
								eliminateValue(s->currentGrid->cells, x, y, s->currentGrid->size, p->letters[0]);
							}
					}
				else // direction = left/right
					{
						s->lastx = x;
						for (s->x1 = 0; s->x1 < s->currentGrid->size; s->x1++) //check above
							{
								if (s->x1 != x)
									{
										s->direction = x > s->x1 ? -1 : 1;
										s->checkValue = 0;
										for (s->offset = 0; s->offset < 3; s->offset++)
											{
												s->value = p->letters[s->offset + 1];
												s->lastx = (s->x1 + (s->offset * s->direction) + s->currentGrid->size) % s->currentGrid->size;
												s->checkValue |= check(s->currentGrid, s->lastx, y, s->value);
												if (s->checkValue == 0)
													{
														s->currentGrid->cells[s->lastx][y].value = s->value;
														eliminateValue(s->currentGrid->cells, s->lastx, y, s->currentGrid->size, s->value);
													}
											}

										//printGrid(currentGrid, x, y);
										if (s->checkValue == 0) //recursive call
											{
												if (s->set == 0)
													{
														s->set = 1;
														//cloneToGrid(currentGrid, res[index]);
														//res[index]->ok = '1';
														cloneToGrid(s->currentGrid, res[s->base]);
														res[s->base]->ok = '1';
													}
												//printGrid(currentGrid, x, y);
												 if(recCount < MAX *2)
												 {
													if (p->next != NULL )
													{
														computeRecursive(s->currentGrid, p->next,nextPath, s->lastx, y, res, recCount);
														//add(&result, &last, temp);
													}
													else
													{
														printf("Next Path %d\n", s->idx );
														p = nextPath[0];
														nextPath++;
														for (s->row = 0; s->row < g->size; s->row++)
														{
															for (s->col = 0; s->col < g->size; s->col++)
																{
																	computeRecursive(s->currentGrid, p,nextPath, s->row, s->col, res, recCount);
																}
														}	
													}
												 }
											}
										//cloneToGrid(previousGrid, currentGrid);
										cloneToGrid(g,s->currentGrid);
										s->currentGrid->cells[x][y].value = p->letters[0];
										eliminateValue(s->currentGrid->cells, x, y, s->currentGrid->size, p->letters[0]);
									}
							}
					}
			}
		}
		else
		{
			printf("Memory Allocation Error");
		}
		/*free(&currentGrid->cells[0]);
		free(currentGrid->cells);
		free(currentGrid);*/
		//return result;
	}
__device__ void add(grid ** base, grid ** last, grid * newList)
	{
		if (newList != NULL)
			{
				if (*base == NULL)
					{
						*base = newList;
						*last = *base;
					}
				else
					{
						(*last)->next = newList;
					}
				while ((*last)->next != NULL)
					{
						*last = (*last)->next;
					}
			}
	}

/*
 __global__ void add( int *a, int *b, int *c )
 {
 int tid = blockIdx.x; // handle the data at this index
 if (tid < N)
 c[tid] = a[tid] + b[tid];
 }*/

int convertUpper(char u)
	{
		int x = (int) u;
		int A = (int) 'A';
		x = x - A;
		return x;
	}
char convertChar(char u)
	{
		int x = (int) u;
		int A = (int) 'A';
		x = x + A;
		return (char) x;
	}

char convert(int x)
	{
		char res = 'a';
		if (x >= 0)
			{
				//int amount = log2((double) x) + (int) res;
				int amount = int(x) + (int) res;

				res = (char) amount;
			}
		else
			{
				res = ' ';
			}
		return res;
	}

__device__ void cloneToGrid(grid * g, grid * g2)
	{
		g2->size = g->size;
		g2->ok = g->ok;
		for (int row = 0; row < g->size; row++)
			{
				for (int col = 0; col < g->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
	}


path * allocate(char c, char c1, char* c2, int direction)
	{
		path *p;
		hipMallocManaged((void **) &p, 1);
		p->next = NULL;
		p->letters[0] = convertUpper(c);
		p->letters[1] = convertUpper(c1);
		p->letters[2] = convertUpper(c2[0]);
		p->letters[3] = convertUpper(c2[1]);
		p->direction = direction;
		p->domain = NULL;
		p->pass = NULL;
		return p;
	}
void printPath(path * p)
	{
		if (p != NULL)
			{
				char dir = p->direction == UP ? 'U' : 'L';
				if (p->domain != NULL)
					{
						printf("[%s]->[%s]\n", p->domain, p->pass);
					}
				int value = (int) p->letters[0];
				if (value > 30)
					{
						printf("[%c]->[%c%c%c]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);
					}
				else
					{
						printf("[%c]->[%c%c%c]%c\n", convertChar(p->letters[0]), convertChar(p->letters[1]), convertChar(p->letters[2]), convertChar(p->letters[3]), dir);
						//printf("[%d]->[%d%d%d]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);

					}
				printPath(p->next);
			}
	}
path * getPath(char * line)
	{
		char * domain = line;
		char * pass = strstr(domain, "-");
		long offset = (long) pass - (long) domain;
		domain[offset - 2] = 0;
		pass += 3;

		int domainLen = strlen(domain);

		//printf("[%s]->[%s]\n", domain, pass);
		path * head = NULL;
		path * tail = NULL;
		char previous = domain[domainLen - 1];
		int direction = domainLen % 2 == 1 ? UP : LEFT;
		for (int i = 0; i < domainLen; i++)
			{
				char next = domain[i];
				path * current = allocate(previous, next, &pass[i * 2], direction);
				previous = pass[i*2+1];
				if (head == NULL)
					{
						head = current;
						hipMallocManaged((void **) &head->domain, domainLen + 1);
						strcpy(head->domain, domain);
						hipMallocManaged((void **) &head->pass, strlen(pass) + 1);
						strcpy(head->pass, pass);
					}
				if (tail == NULL)
					{
						tail = head;
					}
				else
					{
						tail->next = current;
						tail = current;
					}
				direction = direction == UP ? LEFT : UP;
			}

		return head;
	}
path ** scanChars()
	{
		char test[100] = "HEBJCE  -> BJAGDHCHJEGJ";
		int count = 100;
		int index = 0;
		path ** pathList;
		hipMallocManaged((void **) &pathList, (sizeof(path *)) * count);
		path * p = getPath(test);
		printPath(p);
		/*char str[100];
		 scanf("%[^\t\n]99", str);
		 str[99] = 0;
		 */
		FILE * database;
		char buffer[30];

		database = fopen("output.txt", "r");

		if (NULL == database)
			{
				perror("opening database");
				return NULL;
			}
		int max = 10;
		while (EOF != fscanf(database, "%[^\n]\n", buffer) && index < max)
			{
				char * b = buffer;
				while (*b != '-')
					{
						if (*b == 0)
							{
								*b = ' ';
							}
						b++;
					}
				if (index < max)
					{
						p = getPath(buffer);
						pathList[index] = p;
						index++;
						if (index == count)
							{
								path ** temp;
								hipMallocManaged((void **) &temp, (sizeof(path *) * count * 1.5));
								for (int i = 0; i < count - 1; i++)
									{
										temp[i] = pathList[i];
									}
								hipFree(pathList);
								pathList = temp;
								count *= 1.5;

							}

						//printf("> %s\n", buffer);
						//	getPath(buffer);
					}
			}

		/*for (int i = 0; i < count; i++)
		 {
		 printPath(pathList[i]);
		 }*/
		printf("Count is  = %d\n", index);
		fclose(database);
		return pathList;
	}

int foo(path * p, path ** p2)
	{

		hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024); //See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#heap-memory-allocation

	//cell * array;
	//	cell** cells;
		int nBYn = N * N;
		int size = N;
		grid * g = allocateGrid(size);
		/*cudaMallocManaged((void**) &g, sizeof(grid));
		 g->size = N;
		 cudaMallocManaged((void**) &array, nBYn * sizeof(cell));
		 cudaMallocManaged((void **) &cells, size * sizeof(cell *));

		 for (int i = 0; i < size; i++)
		 {
		 cells[i] = &array[i * size];
		 }
		 g->cells = cells;
		 */

		/*	path *p;
		 cudaMallocManaged((void **) &p, 2);
		 path *p2 = p++;


		 p->next = p++;
		 p->direction = UP;
		 p->letters[0] = 4;
		 p->letters[1] = 3;
		 p->letters[2] = 1;
		 p->letters[3] = 4;

		 p2->next = NULL;
		 p2->direction = LEFT;
		 p2->letters[0] = 4;
		 p2->letters[1] = 2;
		 p2->letters[2] = 1;
		 p2->letters[3] = 3;
		 */
		int i = 0;
		grid **result;
		int allocedSize = sizeof(grid*) * size * size * MAX * 2;
		hipMallocManaged((void**) &result, allocedSize);
		printf("Allocated %d bytes %d\n",allocedSize);
		for (int i = 0; i < nBYn * MAX * 2; i++)
			{
				result[i] = allocateGrid(size);
			}
		printf("Done init\n");
		/*for (int row = 0; row < N; row++)
		 {
		 for (int col = 0; col < N; col++)
		 {
		 printf("(%d,,%d)\n", row, col);
		 if (result[i] != NULL)
		 printGrid(result[i], row, col);
		 i++;
		 }
		 }*/
		printPath(p);
		compute<<<size, size>>>(g, p,p2, result);
		hipDeviceSynchronize();
		i = 0;
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{

						for(int j = 0; j <MAX * 2; j+=2)
						{
						int idx = (row * size + col) * MAX*2 +j;
						if (result[idx]->ok == '1')
							{
								printf("(%d,%d,%d)\n", row, col, j);
								printGrid(result[idx]);
							}
						i++;
						}
					}
			}
		/*for( int i = 0; i < nBYn * MAX * 3; i++)
		{
			if (result[i]->ok == '1')
							{
								//printf("(%d,%d,%d)\n", row, col, j);
								puts("");
								printGrid(result[i]);
							}
		}
*/
		//cudaFree(array);
		return 0;
	}

void test2()
	{

	}

/*void initCell(cell * c)
 {

 for (int row = 0; row < N; row++)
 {
 for (int col = 0; col < N; col++)
 {
 c[row * N + col].value = col;
 printf("[%02d][%02d]%02d ", row, col, c[row * N + col].bitmap);
 c[row * N + col].value = -1;
 }
 printf("\n");
 }
 }*/
int main(void)
	{
		path ** p = scanChars();
		if (p != NULL)
			{
				foo(p[1],&p[2]);
			}
	}
/*
 void test1()
 {
 cell * c = (cell *)malloc(N*N*sizeof(cell));
 cell * dev_c;
 cudaMalloc((void **) &dev_c, N*N*sizeof(cell));
 initCell(c);
 cudaMemcpy( dev_c, c, N*N * sizeof(cell), cudaMemcpyHostToDevice) ;
 char ch = 'a';
 ch = (char) (((int) ch) + 7);
 removeIndex<<<10,1>>>( dev_c, 0,5,N,pow(2,7));
 puts("");
 cudaMemcpy( c, dev_c, N*N * sizeof(cell),cudaMemcpyDeviceToHost);
 for (int row=0; row<N; row++)
 {
 for (int col=0; col<N; col++)
 {
 //printf("[%02d][%02d]%02X ",row, col, c[row * N + col].bitmap);
 int index = row * N + col;
 int value = c[index].value;
 char printC = ' ';
 if( value == -1)
 {
 value = c[index].bitmap;
 }
 else

 printC = convert(value);
 printf("%02X-%c ", value, printC);
 }
 printf("\n");
 }
 // free the memory allocated on the GPU
 cudaFree( dev_c );
 }*/
