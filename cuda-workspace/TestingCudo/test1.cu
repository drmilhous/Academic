
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#define N 10
#define UP 'U'
#define LEFT 'L'
#define MAX 9

typedef struct path
	{
	struct path * next;
	char direction;
	int letters[4];
	char * domain;
	char * pass;
	} path;

struct cell
	{
	int value;
	int bitmap;
	};

typedef struct grid
	{
	cell ** cells;
	//cell cells[N][N];
	short size;
	struct grid * next;
	//int xx;
	char ok;
	//int yy;
	} grid;

void initCell(cell * c);
char convert(int x);
__device__ void computeRecursive(grid * g, path * p, path ** nextPath, int x, int y, grid ** res, int recCount);
__device__ void cloneToGrid(grid * g, grid * g2);
__device__ void eliminateValue(cell **c, int row, int col, int max, int value);
__device__ void add(grid ** base, grid ** last, grid * newList);
//void printGrid(grid * g, int x, int y);
__device__ grid * cloneGrid(grid * g);
char convertChar(char u)
	{
		int x = (int) u;
		int A = (int) 'A';
		x = x + A;
		return (char) x;
	}
void printPath(path * p)
	{
		if (p != NULL)
			{
				char dir = p->direction == UP ? 'U' : 'L';
				if (p->domain != NULL)
					{
						printf("[%s]->[%s]\n", p->domain, p->pass);
					}
				int value = (int) p->letters[0];
				if (value > 30)
					{
						printf("[%c]->[%c%c%c]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);
					}
				else
					{
						printf("[%c]->[%c%c%c]%c\n", convertChar(p->letters[0]), convertChar(p->letters[1]), convertChar(p->letters[2]), convertChar(p->letters[3]), dir);
						//printf("[%d]->[%d%d%d]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);

					}
				printPath(p->next);
			}
	}
__device__ char convertChar2(char u)
	{
		int x = (int) u;
		int A = (int) 'A';
		x = x + A;
		return (char) x;
	}
__device__ void printPath2(path * p)
	{
		printf("Pointer %p ",p);
		if (p != NULL)
			{
				
				char dir = p->direction == UP ? 'U' : 'L';
				if (p->domain != NULL)
					{
						printf("[%s]->[%s]\n", p->domain, p->pass);
					}
				int value = (int) p->letters[0];
				if (value > 30)
					{
						printf("[%c]->[%c%c%c]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);
					}
				else
					{
						printf("[%c]->[%c%c%c]%c\n", convertChar2(p->letters[0]), convertChar2(p->letters[1]), convertChar2(p->letters[2]), convertChar2(p->letters[3]), dir);
						//printf("[%d]->[%d%d%d]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);

					}
				printPath2(p->next);
			}
	}

void printGrid(grid * g)
	{
		int n = g->size;
		//printf("X=%d Y=%d %c\n", x, y, g->ok);
		for (int row = 0; row < n; row++)
			{
				for (int col = 0; col < n; col++)
					{
						cell c = g->cells[row][col];
						//printf("[%02d][%02d]%02X ",row, col, c[row * N + col].bitmap);
						int value = c.value;
						char printC = ' ';
						if (value < 0)
							{
								value = c.bitmap;
								printf(" %03X%c", value, printC);
							}
						else
							{
								//
								printC = convert(value);
								value = 0;
								printf("  %c  ", printC);
							}
						
					}
				printf("\n");
			}
	}
	__device__ int pow2(int x)
{
	int sum = 1;
	if( x == 0)
	{
		sum = 1;
	}
	else
	{
	for(int i = 0; i < x; i++)
	{
		sum = sum *2;
	}
	}
	return sum;
}
__device__ void eliminateValue(cell **c, int row, int col, int max, int value)
	{
		//int mask = pow(2.0, (double) value);
		int mask = pow2(value);
		for (int r1 = 0; r1 < max; r1++)
			{
				if (r1 != row)
					{
						c[r1][col].bitmap |= mask;
					}
			}
		for (int c1 = 0; c1 < max; c1++)
			{
				if (c1 != col)
					{
						c[row][c1].bitmap |= mask;
					}
				//x->c = ch;
			}
	}

__device__ int check(grid * g, int row, int col, int number)
	{
		int result;
		cell * c = &g->cells[row][col];
		if (c->value >= 0 && c->value != number)
			{
				result = 1;
			}
		else
			{
				//int mask = pow(2.0, (double) number);
				int mask = pow2(number);
				int bits = c->bitmap;
				result = (mask & bits);
				if(result != 0)
				{
					result = 1;
				}
			}
		return result;
	}
grid * allocateGrid(int size)
	{

		grid * g2 = NULL;
		hipMallocManaged((void **) &g2, sizeof(grid));
		g2->size = size;
		//cell * array;
		//cudaMallocManaged((void **) &array, size * size * sizeof(cell));
		cell ** cells;
		hipMallocManaged((void **) &cells, size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				//cells[i] = &array[i * size];
				hipMallocManaged((void **) &cells[i], size * sizeof(cell));
			}
		g2->cells = cells;

		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		//printf("XX%p->%c\n",g2, g2->ok);
		return g2;
	}
__device__ grid * cloneGrid(grid * g)
	{
		grid * g2 = (grid *) malloc(sizeof(grid));
		if(g2 != NULL)
		{
		g2->size = g->size;
		cell * array = (cell *) malloc(g->size * g2->size * sizeof(cell));
		cell ** cells = (cell **) malloc(g2->size * sizeof(cell *));
		for (int i = 0; i < g->size; i++)
			{
				cells[i] = &array[i * g2->size];
			}
		g2->cells = cells;
		for (int row = 0; row < g2->size; row++)
			{
				for (int col = 0; col < g2->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		}
		return g2;
	}
__global__ void compute(grid * g, path * p,path ** p2, grid ** result)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < N * N)
			{
				int x = blockIdx.x;
				int y = threadIdx.x;
				computeRecursive(g, p,p2, x, y, result, 0);
				//printPath2(p);
				//p2++;
				//printPath2(p2[0]);
			}
	}
__device__ void computeRecursive(grid * g, path * p, path ** nextPath, int x, int y, grid ** res, int recCount)
{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int base = idx * MAX *3 + recCount;
		uint8_t rec = (p->next == NULL);
		if(rec == 1)
		{
			p = nextPath[0];
			nextPath++;
		}
		//printf("index[%02d] base[%d]\n",idx, base);
		grid * currentGrid = res[base +1];
		recCount = recCount +3 ;
		uint8_t checkValue = 0;
		res[base]->ok= '0';
		uint8_t value = p->letters[0];
		//grid * currentGrid = cloneGrid(g);
		cloneToGrid(g,currentGrid);
		if(currentGrid != NULL)
		{
		grid* previousGrid = res[base + 2];
		checkValue = check(currentGrid, x, y, value);
		if (checkValue == 0)
			{
				currentGrid->cells[x][y].value = value;
				eliminateValue(currentGrid->cells, x, y, currentGrid->size, value);
				cloneToGrid(currentGrid, previousGrid);
				if (p->direction == LEFT) //Do UP/DOWN
					{
						int8_t lasty = y;
						for (uint8_t y1 = 0; y1 < currentGrid->size; y1++) //check above
							{
								if (y1 != y)
									{
										int8_t direction = y > y1 ? -1 : 1;
										checkValue = 0;
										for (uint8_t offset = 0; offset < 3; offset++)
											{
												value = p->letters[offset + 1];
												lasty = (y1 + (offset * direction) + currentGrid->size) % currentGrid->size;
												checkValue |= check(currentGrid, x, lasty, value);
												if (checkValue == 0)
													{
														currentGrid->cells[x][lasty].value = value;
														eliminateValue(currentGrid->cells, x, lasty, currentGrid->size, value);
													}
											}
										if (checkValue == 0) //recursive call
											{
												if (res[base]->ok == '0')
													{
														cloneToGrid(currentGrid, res[base]);
														res[base]->ok = '1';
													}
												//if (p->next != NULL && recCount < MAX * 3)
												//	{
												//		computeRecursive(currentGrid, p->next, x, lasty, res, recCount);
												//	}
												if(recCount < MAX *3)
												 {
													if (rec == 0)
													{
														computeRecursive( currentGrid, p->next,nextPath,  x, lasty, res, recCount);
													}
													else
													{
														//printf("Next Path %d\n",  blockIdx.x * blockDim.x + threadIdx.x );
														//p = nextPath[0];
														//nextPath++;
														for (uint8_t row = 0;  row < g->size;  row++)
														{
															for (uint8_t col = 0;  col < g->size;  col++)
																{
																	computeRecursive( currentGrid, p,nextPath,  row,  col, res, recCount);
																}
														}	
													}
												 }
											}
									}
								cloneToGrid(previousGrid, currentGrid);
							}
					}
				else // direction = left/right
					{
						int8_t lastx = x;
						for (uint8_t x1 = 0; x1 < currentGrid->size; x1++) //check above
							{
								if (x1 != x)
									{
										int8_t direction = x > x1 ? -1 : 1;
										checkValue = 0;
										for (uint8_t offset = 0; offset < 3; offset++)
											{
												value = p->letters[offset + 1];
												lastx = (x1 + (offset * direction) + currentGrid->size) % currentGrid->size;
												checkValue |= check(currentGrid, lastx, y, value);
												if (checkValue == 0)
													{
														currentGrid->cells[lastx][y].value = value;
														eliminateValue(currentGrid->cells, lastx, y, currentGrid->size, value);
													}
											}

										//printGrid(currentGrid, x, y);
										if (checkValue == 0) //recursive call
											{
												if (res[base]->ok == '0')
													{
														cloneToGrid(currentGrid, res[base]);
														res[base]->ok = '1';
													}
												//if (p->next != NULL && recCount < MAX *3)
												//	{
												//	computeRecursive(currentGrid, p->next, lastx, y, res, recCount);
												//	}
												if(recCount < MAX *3)
												 {
													if (rec == 0)
													{
														computeRecursive( currentGrid, p->next,nextPath,  lastx, y, res, recCount);
													}
													else
													{
														printf("Next Path %d\n",  blockIdx.x * blockDim.x + threadIdx.x );

														//printf("p = %p, next = %p\n", p, nextPath[0]);
														
														//printf("p = %p, next = %pX\n", p, &nextPath[0]);
														//printPath2(nextPath[0]);
														for (uint8_t row = 0;  row < g->size;  row++)
														{
															for (uint8_t col = 0;  col < g->size;  col++)
																{
																	computeRecursive( currentGrid, p,nextPath,  row,  col, res, recCount);
																}
														}	
													}
												 }
											}
										cloneToGrid(previousGrid, currentGrid);
									}
							}
					}
			}
		}
		else
		{
			printf("Memory Allocation Error");
		}
		/*free(&currentGrid->cells[0]);
		free(currentGrid->cells);
		free(currentGrid);*/
		//return result;
	}
__device__ void add(grid ** base, grid ** last, grid * newList)
	{
		if (newList != NULL)
			{
				if (*base == NULL)
					{
						*base = newList;
						*last = *base;
					}
				else
					{
						(*last)->next = newList;
					}
				while ((*last)->next != NULL)
					{
						*last = (*last)->next;
					}
			}
	}

/*
 __global__ void add( int *a, int *b, int *c )
 {
 int tid = blockIdx.x; // handle the data at this index
 if (tid < N)
 c[tid] = a[tid] + b[tid];
 }*/

int convertUpper(char u)
	{
		int x = (int) u;
		int A = (int) 'A';
		x = x - A;
		return x;
	}


char convert(int x)
	{
		char res = 'a';
		if (x >= 0)
			{
				//int amount = log2((double) x) + (int) res;
				int amount = int(x) + (int) res;

				res = (char) amount;
			}
		else
			{
				res = ' ';
			}
		return res;
	}

__device__ void cloneToGrid(grid * g, grid * g2)
	{
		g2->size = g->size;
		g2->ok = g->ok;
		for (int row = 0; row < g->size; row++)
			{
				for (int col = 0; col < g->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
	}


path * allocate(char c, char c1, char* c2, int direction)
	{
		path *p;
		hipMallocManaged((void **) &p, 1 * sizeof(path));
		p->next = NULL;
		p->letters[0] = convertUpper(c);
		p->letters[1] = convertUpper(c1);
		p->letters[2] = convertUpper(c2[0]);
		p->letters[3] = convertUpper(c2[1]);
		p->direction = direction;
		p->domain = NULL;
		p->pass = NULL;
		return p;
	}

path * getPath(char * line)
	{
		char * domain = line;
		char * pass = strstr(domain, "-");
		long offset = (long) pass - (long) domain;
		domain[offset - 2] = 0;
		pass += 3;

		int domainLen = strlen(domain);

		//printf("[%s]->[%s]\n", domain, pass);
		path * head = NULL;
		path * tail = NULL;
		char previous = domain[domainLen - 1];
		int direction = domainLen % 2 == 1 ? UP : LEFT;
		for (int i = 0; i < domainLen; i++)
			{
				char next = domain[i];
				path * current = allocate(previous, next, &pass[i * 2], direction);
				previous = pass[i*2+1];
				if (head == NULL)
					{
						head = current;
						hipMallocManaged((void **) &head->domain, domainLen + 1);
						strcpy(head->domain, domain);
						hipMallocManaged((void **) &head->pass, strlen(pass) + 1);
						strcpy(head->pass, pass);
					}
				if (tail == NULL)
					{
						tail = head;
					}
				else
					{
						tail->next = current;
						tail = current;
					}
				direction = direction == UP ? LEFT : UP;
			}

		return head;
	}
path ** scanChars()
	{
		//char test[100] = "HEBJCE  -> BJAGDHCHJEGJ";
		int count = 100;
		int index = 0;
		path ** pathList;
		hipMallocManaged((void **) &pathList, (sizeof(path *)) * count * 2);
		//path * p = getPath(test);
		path * p;
		//printPath(p);
		/*char str[100];
		 scanf("%[^\t\n]99", str);
		 str[99] = 0;
		 */
		FILE * database;
		char buffer[30];

		database = fopen("output.txt", "r");

		if (NULL == database)
			{
				perror("opening database");
				return NULL;
			}
		int max = 100;
		while (EOF != fscanf(database, "%[^\n]\n", buffer) && index < max)
			{
				char * b = buffer;
				while (*b != '-')
					{
						if (*b == 0)
							{
								*b = ' ';
							}
						b++;
					}
				if (index < max)
					{
						p = getPath(buffer);
						pathList[index] = p;
						index++;
						if (index == count)
							{
								path ** temp;
								hipMallocManaged((void **) &temp, (sizeof(path *) * count * 1.5));
								for (int i = 0; i < count - 1; i++)
									{
										temp[i] = pathList[i];
									}
								hipDeviceSynchronize();
								hipFree(pathList);
								pathList = temp;
								count *= 1.5;

							}

						//printf("> %s\n", buffer);
						//	getPath(buffer);
					}
			}

		for (int i = 0; i < count; i++)
		 {
		 
		 	printPath(pathList[i]);
		 }

		printf("\nCount is  = %d\n", index);
		fclose(database);
		return pathList;
	}
int foo(path * p, path ** p2)
	{

		hipDeviceSetLimit(hipLimitMallocHeapSize, 128 * 1024 * 1024 * 8); //See more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#heap-memory-allocation

	//cell * array;
	//	cell** cells;
		int nBYn = N * N;
		int size = N;
		grid * g = allocateGrid(size);
		/*cudaMallocManaged((void**) &g, sizeof(grid));
		 g->size = N;
		 cudaMallocManaged((void**) &array, nBYn * sizeof(cell));
		 cudaMallocManaged((void **) &cells, size * sizeof(cell *));

		 for (int i = 0; i < size; i++)
		 {
		 cells[i] = &array[i * size];
		 }
		 g->cells = cells;
		 */

		/*	path *p;
		 cudaMallocManaged((void **) &p, 2);
		 path *p2 = p++;


		 p->next = p++;
		 p->direction = UP;
		 p->letters[0] = 4;
		 p->letters[1] = 3;
		 p->letters[2] = 1;
		 p->letters[3] = 4;

		 p2->next = NULL;
		 p2->direction = LEFT;
		 p2->letters[0] = 4;
		 p2->letters[1] = 2;
		 p2->letters[2] = 1;
		 p2->letters[3] = 3;
		 */
		int i = 0;
		grid **result;
		int allocedSize = sizeof(grid*) * size * size * MAX * (4);
		hipMallocManaged((void**) &result, allocedSize);
		printf("Allocated %d bytes\n",allocedSize);
		for (int i = 0; i < nBYn * MAX * 4; i++)
			{
				result[i] = allocateGrid(size);
			}
		printf("Done init\n");
		/*for (int row = 0; row < N; row++)
		 {
		 for (int col = 0; col < N; col++)
		 {
		 printf("(%d,,%d)\n", row, col);
		 if (result[i] != NULL)
		 printGrid(result[i], row, col);
		 i++;
		 }
		 }*/
		//printPath(p);
			hipDeviceSynchronize();
		printPath(&p[0]);
		printPath(&p[1]);
			
		compute<<<size, size>>>(g, p,p2, result);
		hipDeviceSynchronize();
		i = 0;
		for (int row = 0; row < N; row++)
			{
				for (int col = 0; col < N; col++)
					{

						for(int j = 0; j <MAX * 3; j+=3)
						{
						int idx = (row * size + col) * MAX*3 +j;
						if (result[idx]->ok == '1')
							{
								printf("(%d,%d,%d)\n", row, col, j);
								printGrid(result[idx]);
							}
						i++;
						}
					}
			}
		/*for( int i = 0; i < nBYn * MAX * 3; i++)
		{
			if (result[i]->ok == '1')
							{
								//printf("(%d,%d,%d)\n", row, col, j);
								puts("");
								printGrid(result[i]);
							}
		}
*/
		//cudaFree(array);
		return 0;
	}
void test2()
	{

	}

/*void initCell(cell * c)
 {

 for (int row = 0; row < N; row++)
 {
 for (int col = 0; col < N; col++)
 {
 c[row * N + col].value = col;
 printf("[%02d][%02d]%02d ", row, col, c[row * N + col].bitmap);
 c[row * N + col].value = -1;
 }
 printf("\n");
 }
 }*/
int main(void)
	{
		path ** p = scanChars();
		if (p != NULL)
			{
				foo(p[0], &p[0]);
			}
	}
/*
 void test1()
 {
 cell * c = (cell *)malloc(N*N*sizeof(cell));
 cell * dev_c;
 cudaMalloc((void **) &dev_c, N*N*sizeof(cell));
 initCell(c);
 cudaMemcpy( dev_c, c, N*N * sizeof(cell), cudaMemcpyHostToDevice) ;
 char ch = 'a';
 ch = (char) (((int) ch) + 7);
 removeIndex<<<10,1>>>( dev_c, 0,5,N,pow(2,7));
 puts("");
 cudaMemcpy( c, dev_c, N*N * sizeof(cell),cudaMemcpyDeviceToHost);
 for (int row=0; row<N; row++)
 {
 for (int col=0; col<N; col++)
 {
 //printf("[%02d][%02d]%02X ",row, col, c[row * N + col].bitmap);
 int index = row * N + col;
 int value = c[index].value;
 char printC = ' ';
 if( value == -1)
 {
 value = c[index].bitmap;
 }
 else

 printC = convert(value);
 printf("%02X-%c ", value, printC);
 }
 printf("\n");
 }
 // free the memory allocated on the GPU
 cudaFree( dev_c );
 }*/
