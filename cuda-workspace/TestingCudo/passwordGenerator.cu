#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 1024

//making my own strcpy and and str cat because screw cuda, not giving access to libraries :(
__device__ char* nStrCpy(char *dest, const char *src) {
	int i =0;
	do {
		dest[i] = src[1];
	} while (src[i++] != 0);
	return dest;
}

__device__ char* nStrcat(char *dest, const char *src){
	int i =0;
	while (dest[i] != 0) i++;
	nStrCpy(dest+1, src);
	return dest;
}

//this makes a single password, recursivly adding 2 characters to password every time and removing one from site
__device__ void  makePassword(char *square, char* site, int position, int direction, int size, char* password) {
	//x position and y position within square as square is a linear array
	int x = position%size;
	int y = position/size;
	int firstCharP = 0;
	int secCharP = 0;

	//if direction is vertical
	if (direction ==0) {
		//check every character in the current vertical line
		for (int i =0; i < size; i++) {

			//position of new character
			int newPosition = (i*size) + x;

			//found a match
			if (site[0] == square[newPosition]) {
				//goes up
				if (newPosition < position) {
					//first character for password
					firstCharP = newPosition - size;
					//if below first line go to bottom
					if(firstCharP < 0) 
						firstCharP += (size * size);

					//second character for password
					secCharP = firstCharP - size;
					if(secCharP < 0)
						secCharP += (size*size);
				//goes down
				} else {
					firstCharP = newPosition + size;
					// if below last line, loop to top
					if (firstCharP >= (size*size))
						firstCharP -= (size*size);
					
					secCharP = firstCharP + size;
					if(secCharP >= (size*size))
						secCharP -= (size*size);
				}
			}
		}
		//switch to horizontal directiuon for next 2 characters
		direction = 1;

	//if direction is horizontal
	} else {
		for (int i =0; i < size; i++) {
			int newPosition = (y*size)+i;
			if (site[0] == square[newPosition]) {
				//new position to the left of previous, should never be the same
				if (newPosition < position) {
					firstCharP = newPosition -1;
					//if previous row, wrap around to right side instead
					if ((firstCharP/size) < y || firstCharP == -1)
						firstCharP += size;
					
					secCharP = firstCharP -1;
					if ((secCharP/size) < y || secCharP == -1)
						secCharP += size;
				//new position to right of previous
				} else {
					//if on next row wrap to front
					firstCharP  = newPosition +1;
					if ((firstCharP/size) > y)
						firstCharP -= size;

				secCharP = firstCharP +1;
					if ((secCharP/size) > y)
						secCharP -= size;
				}
			}
		}
		//switch to vertical direction for next couple of characters
		direction = 0;
	}
	
	//go to next character in site name
	site++;
	//if more of the password is neeeded
	if (site[0] != '\n') {
		//set the next couple of characters
		password[0] = square[firstCharP];
		password[1] = square[secCharP];
		//increase pointer to start the next part of password without overwrting previous characters
		password++;
		password++;
		
		//mor parts of the password!
		makePassword(square, site, secCharP, direction, size, password);
	} else {
		//set the last two characters of the password.
		password[0] = square[firstCharP];
		password[1] = square[secCharP];
	}
}

//get the starting poisition of the password within the gride, i.e. start at top row, and travel through the domain name
__device__ int getStartPosition(char *square, char *site, int size) {
	int position =0;

	//find the atarting position within the first row
	for (int i =0; i < size; i++) {
		if (square[i]  == site[0])
			position = i;
	}

	//direction 0 is going down, as it starts	
	int direction = 0;
	
	//doing 6 characters only, because apparently I hate make modularized code the first time
	for (int i =1; i < 6; i++) {

		//x and y position within a linear array
		int x = position%size;
		int y = position/size;

		//check all characters in row/colums
		for (int j = 0; j < size; j++) {
			//vertical directions
			if (direction ==0) {
				//it found the next character!
				if (site[i] == square[(j * size) + x ]) {
					position = (j * size) +x;
					direction = 1;
					break;
				}
			//horizontal direction
			} else {
				//it found the nest character!
				if (site[i] == square[(y * size) + j]) {
					position = (y* size) + j;
					direction = 0;
					break;
				}
			}
		}
	}
	//return the starting poistion... because that's the point of this function ... dur
	return position;
}

//make a random password
__global__ void randomWords(char *square, char *passwords, int size, int *c, int amount) {
	//that id though
	int tid = blockIdx.x*blockDim.x+threadIdx.x;

	//cuda random intitalizers
	hiprandState_t state;
	hiprand_init(tid, 1, 2, &state);
	
	//make a certain number of passwords per core
	for (int a = 0; a < amount; a++) {

		//starting position for this password
		int tidNum = ((tid * amount) + a) *24;
		passwords[(tidNum)] = square[(hiprand(&state) % size)];
		
		//7 characters for the site, 6 and a \n
		char site[7];
		site[0] = passwords[tidNum];
		site[6] = '\n';

		//make 6 random characters
		for (int i=1; i < 6; i++) {
			//make sure 2 characters do not repeat
			do {
			passwords[i + (tidNum)] = square[(hiprand(&state) % size)];
			} while (passwords[(i-1) +(tidNum)] == passwords[i + (tidNum)]);

			//set random character
			site[i] = passwords[i + (tidNum)];
		}

		// add that ' -> ' Miller wanted
		passwords[7 + (tidNum)] = ' ';
		passwords[8 + (tidNum)] = '-';
		passwords[9 + (tidNum)] = '>';
		passwords[10 + (tidNum)] = ' ';

		//lets get that starting position
		int position = getStartPosition(square, site, size);
		
		//stored the startingposition within c for debuggin puroposes
		//I left this in here becuase it could be useful if I ever come back to this project
		c[(tid * amount)+ a] = position;

		//create the password object
		char *password;
		password = (char *)malloc(sizeof(char) *13);

		//generate that password finally
		makePassword(square, site, position, 1, size, password);

		//save the password in the passwords array that the main program can access
		for(int i = 0; i < 12; i++) {
			passwords[11 + i + (tidNum)] = password[i];
		}
	}
}

int main(int argc, char ** argv) 
{	
	//used to organize cores on cuda
	dim3 gridsize, blocksize;
	int device = atoi(argv[1]);
	hipSetDevice(device);
	//get size of the grid
	int size = 15;
	//printf("Please input a size of grid to be tested (integer number only): ");
	//scanf("%d", &size);

	//I want at least total passwords (I used 12,000 because when creating 10,000 passwords
	//out of a possible 100,000 there are bound to be some repeats
	int total = 500;
	int amount = total / N;
	amount++;
	total = amount * N;

	//get the file to be read
	char file[] = "grid15.txt";
	//printf("Insert a file containing your latin square: ");
	//scanf("%s", file);

	//allocate memory for the grid
	char grid[size][size];
	char *square;
	hipMallocManaged((void**)&square, size * size * sizeof(char));
	
	//allocate memory for the grid
	char *passwords;
	printf("total: %d\n", total);
	hipMallocManaged((void**)&passwords, sizeof(char) * 24 * total);
	
	//open grid file to read grid
	FILE *file1 = fopen(file, "r");

	//copy each character from grid file to grid object
	for (int i=0; i < size; i++) {
		for (int j=0; j < size; j++) {
			fscanf(file1, "%c\n", &grid[i][j]);
		}
		
	}

	//transfer the grid to a linear array
	for(int i=0; i < size;i++) {
		for(int j=0; j < size; j++) {
			square[size * i + j] = grid[i][j];
		}
	}

	//close the grid file
	fclose(file1);

	// allocate the memory on the GPU, this was used for saving the starting positions of each password
	int *c;
	hipMallocManaged( (void**)&c, N * amount * sizeof(int));

	//I randomly chose 16 as the block size, it seems like a good number
	blocksize.x = 16;
	gridsize.x = N/blocksize.x;
 
	//this activates some cool cuda stuff
	randomWords<<<gridsize.x, blocksize.x>>>(square, passwords, size, c, amount);
	hipDeviceSynchronize();
	
	//outpt file brah
	FILE * f = fopen("output15.txt", "w");

	//lets make sure that file exists brh
	if (f == NULL) {
		printf("error opening output.txt\n");
		exit(1);
	}

	//copy the passwords to the file one character at a time. oh yeah the effeciency broseph
	for (int i=0; i<total; i++)
	 {
		char * output = (char *)malloc(sizeof(char) * 24);
		for (int j=0; j<23;j++) {
			fprintf(f, "%c", passwords[j + (i * 24)]);
		}
		fprintf(f, "\n");
	}


	// free the memory allocated on the GPU, close the file and you are done Tyranbrosaurus Rex!
	fclose(f);
	hipFree( c );
	hipFree( square );
	hipFree( passwords );
	return 0;
}
