
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 10
/*
typedef struct graph{

	char * data;
	char direction;
}graph;

__device__ graph * graphAllocate()
{
	graph* g = (graph *) malloc(sizeof(graph));
	g->data = (char *)malloc(1000);
	for(int i = 0; i < 1000;i++)
	{
		g->data[i] = 254;
	}
	g->direction = 'M';
	printf("Before '%c' ", g->direction);
	return g;
}
__device__ void b(int idx)
{
	graph * g = graphAllocate();
	int sum = 1;
	for(int i = 0; i < 1000;i++)
	{
		sum = (sum + g->data[i]) % idx;
	}
	printf("After'%c' %02d\n", g->direction,sum);
	//printf("Device %d\n", idx);
}

__global__ void a()
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N * N)
	{
		b(idx);
	}
}

/*
int main()
{
	//cout << "Yo Yo" << endl;
	printf("Hii\n");
	a<<<N,N>>>();
	cudaDeviceSynchronize();
	printf("Yo\n");
	return 0;
}*/

__device__ int  recursive(int count, int * t)
{
	int res = 0;
	count--;
	if(count > 0)
	{
		int *x = (int *)malloc(count);
		*x = count;
		res += recursive(count, x);
		free(x);
	}
	else
	{
		res = count+1;
	}
	return res + *t;
}
__global__ void rec(int count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N * N)
	{
		int * temp = (int *) malloc(1);
		*temp = recursive(count, temp);
		printf("Sume = %d", *temp);
	}
}
int main()
{
	int value;
	printf("Enter the size:");
	scanf("%d", &value);
	//cout << "Yo Yo" << endl;
	printf("Hii\n");
	rec<<<N,N>>>(value);
	hipDeviceSynchronize();
	printf("Yo\n");
	return 0;
}



