#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N 10

typedef struct graph{

	char * data;
	char direction;
}graph;

__device__ graph * graphAllocate()
{
	graph* g = (graph *) malloc(sizeof(graph));
	g->data = (char *)malloc(1000);
	for(int i = 0; i < 1000;i++)
	{
		g->data[i] = 254;
	}
	g->direction = 'M';
	printf("Before '%c' ", g->direction);
	return g;
}
__device__ void b(int idx)
{
	graph * g = graphAllocate();
	int sum = 1;
	for(int i = 0; i < 1000;i++)
	{
		sum = (sum + g->data[i]) % idx;
	}
	printf("After'%c' %02d\n", g->direction,sum);
	//printf("Device %d\n", idx);
}

__global__ void a()
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N * N)
	{
		b(idx);
	}
}

/*
int main()
{
	//cout << "Yo Yo" << endl;
	printf("Hii\n");
	a<<<N,N>>>();
	hipDeviceSynchronize();
	printf("Yo\n");
	return 0;
}*/

int main()
{
	int value;
	printf("Enter the size:");
	scanf("%d", &value);
	//cout << "Yo Yo" << endl;
	printf("Hii\n");
	rec<<<N,N>>>(value);
	hipDeviceSynchronize();
	printf("Yo\n");
	return 0;
}

__device__ void recursive(int count)
{
	count--;
	if(count > 0)
	{
		recursive(count);
	}
}
__global__ void rec(int count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N * N)
	{
		recursive(count);
	}
}

