
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 10
/*
typedef struct graph{

	char * data;
	char direction;
}graph;

__device__ graph * graphAllocate()
{
	graph* g = (graph *) malloc(sizeof(graph));
	g->data = (char *)malloc(1000);
	for(int i = 0; i < 1000;i++)
	{
		g->data[i] = 254;
	}
	g->direction = 'M';
	printf("Before '%c' ", g->direction);
	return g;
}
__device__ void b(int idx)
{
	graph * g = graphAllocate();
	int sum = 1;
	for(int i = 0; i < 1000;i++)
	{
		sum = (sum + g->data[i]) % idx;
	}
	printf("After'%c' %02d\n", g->direction,sum);
	//printf("Device %d\n", idx);
}

__global__ void a()
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N * N)
	{
		b(idx);
	}
}

/*
int main()
{
	//cout << "Yo Yo" << endl;
	printf("Hii\n");
	a<<<N,N>>>();
	cudaDeviceSynchronize();
	printf("Yo\n");
	return 0;
}*/

__device__ int  recursive(int all, int count, int * t)
{
	int res = 0;
	count--;
	if(count > 0)
	{
		int *x = (int *)malloc(all);
		*x = count;
		res += recursive(all, count, x);
		free(x);
	}
	else
	{
		res = count+1;
	}
	return res + *t;
}
__global__ void rec(int count, int allocate)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//if(idx < N * N)
	{
		int * temp = (int *) malloc(allocate);
		*temp = 0;
		*temp = recursive(allocate, count, temp);
		printf("Sume = %d\n", *temp);
	}
}
/*int main()
{
	int value;
	int allocate;
	printf("Enter the size:");
	scanf("%d", &value);
	printf("Enter the Stack size:");
	scanf("%d", &allocate);
	//cout << "Yo Yo" << endl;
	printf("Hii\n");
	rec<<<N,N>>>(value, allocate);
	cudaDeviceSynchronize();
	printf("Yo\n");
	return 0;
}



*/