#include "hip/hip_runtime.h"
#include "grid.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
path ** scanChars()
	{
		char test[100] = "HEBJCE  -> BJAGDHCHJEGJ";
		int count = 100;
		int index = 0;
		path ** pathList;
		hipMallocManaged((void **) &pathList, (sizeof(path *)) * count);
		path * p = getPath(test);
		printPath(p);
		FILE * database;
		char buffer[30];

		database = fopen("output.txt", "r");

		if (NULL == database)
			{
				perror("opening database");
				return NULL;
			}
		int max = 10;
		while (EOF != fscanf(database, "%[^\n]\n", buffer) && index < max)
			{
				char * b = buffer;
				while (*b != '-')
					{
						if (*b == 0)
							{
								*b = ' ';
							}
						b++;
					}
				if (index < max)
					{
						p = getPath(buffer);
						pathList[index] = p;
						index++;
						if (index == count)
							{
								path ** temp;
								hipMallocManaged((void **) &temp, (sizeof(path *) * count * 1.5));
								for (int i = 0; i < count - 1; i++)
									{
										temp[i] = pathList[i];
									}
								hipFree(pathList);
								pathList = temp;
								count *= 1.5;

							}

						//printf("> %s\n", buffer);
						//	getPath(buffer);
					}
			}

		/*for (int i = 0; i < count; i++)
		 {
		 printPath(pathList[i]);
		 }*/
		printf("Count is  = %d\n", index);
		fclose(database);
		return pathList;
	}
path * allocate(char c, char c1, char* c2, int direction)
	{
		path *p;
		hipMallocManaged((void **) &p, 1);
		p->next = NULL;
		p->letters[0] = convertUpper(c);
		p->letters[1] = convertUpper(c1);
		p->letters[2] = convertUpper(c2[0]);
		p->letters[3] = convertUpper(c2[1]);
		p->direction = direction;
		p->domain = NULL;
		p->pass = NULL;
		return p;
	}
__device__ int pow2(int x)
{
	int sum = 1;
	if( x == 0)
	{
		sum = 1;
	}
	else
	{
	for(int i = 0; i < x; i++)
	{
		sum = sum *2;
	}
	}
	return sum;
}
__device__ void add(grid ** base, grid ** last, grid * newList)
	{
		if (newList != NULL)
			{
				if (*base == NULL)
					{
						*base = newList;
						*last = *base;
					}
				else
					{
						(*last)->next = newList;
					}
				while ((*last)->next != NULL)
					{
						*last = (*last)->next;
					}
			}
	}

/*
 __global__ void add( int *a, int *b, int *c )
 {
 int tid = blockIdx.x; // handle the data at this index
 if (tid < N)
 c[tid] = a[tid] + b[tid];
 }*/

int convertUpper(char u)
	{
		int x = (int) u;
		int A = (int) 'A';
		x = x - A;
		return x;
	}
char convertChar(char u)
	{
		int x = (int) u;
		int A = (int) 'A';
		x = x + A;
		return (char) x;
	}

__device__ char convert(int x)
	{
		char res = 'a';
		if (x >= 0)
			{
				//int amount = log2((double) x) + (int) res;
				int amount = int(x) + (int) res;

				res = (char) amount;
			}
		else
			{
				res = ' ';
			}
		return res;
	}

__device__ void cloneToGrid(grid * g, grid * g2)
	{
		g2->size = g->size;
		g2->ok = g->ok;
		for (int row = 0; row < g->size; row++)
			{
				for (int col = 0; col < g->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
	}


	grid * allocateGrid(int size)
	{
		grid * g2 = NULL;
		hipMallocManaged((void **) &g2, sizeof(grid));
		g2->size = size;
		cell ** cells;
		hipMallocManaged((void **) &cells, size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				hipMallocManaged((void **) &cells[i], size * sizeof(cell));
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}

__device__	grid * allocateGridDevice(int size)
	{
		grid * g2 = NULL;
		g2 = (grid *) malloc(sizeof(grid));
		g2->size = size;
		cell ** cells;
		cells = (cell **) malloc(size * sizeof(cell *));
		for (int i = 0; i < size; i++)
			{
				cells[i] = (cell*) malloc(size * sizeof(cell));
			}
		g2->cells = cells;
		for (int row = 0; row < size; row++)
			{
				for (int col = 0; col < size; col++)
					{
						g2->cells[row][col].bitmap = 0;
						g2->cells[row][col].value = -1;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		return g2;
	}

__device__ grid * cloneGrid(grid * g)
	{
		grid * g2 = (grid *) malloc(sizeof(grid));
		if(g2 != NULL)
		{
		g2->size = g->size;
		cell * array = (cell *) malloc(g->size * g2->size * sizeof(cell));
		cell ** cells = (cell **) malloc(g2->size * sizeof(cell *));
		for (int i = 0; i < g->size; i++)
			{
				cells[i] = &array[i * g2->size];
			}
		g2->cells = cells;
		for (int row = 0; row < g2->size; row++)
			{
				for (int col = 0; col < g2->size; col++)
					{
						g2->cells[row][col].bitmap = g->cells[row][col].bitmap;
						g2->cells[row][col].value = g->cells[row][col].value;
					}
			}
		g2->next = NULL;
		g2->ok = '0';
		}
		return g2;
	}

__device__ void eliminateValue(cell **c, int row, int col, int max, int value)
	{
		int mask = pow2(value);
		for (int r1 = 0; r1 < max; r1++)
			{
				if (r1 != row)
					{
						c[r1][col].bitmap |= mask;
					}
			}
		for (int c1 = 0; c1 < max; c1++)
			{
				if (c1 != col)
					{
						c[row][c1].bitmap |= mask;
					}
			}
	}

__device__ int check(grid * g, int row, int col, int number)
	{
		int result;
		cell * c = &g->cells[row][col];
		if (c->value >= 0 && c->value != number)
			{
				result = 1;
			}
		else
			{
				int mask = pow2(number);
				int bits = c->bitmap;
				result = (mask & bits);
			}
		return result;
	}

path * getPath(char * line)
	{
		char * domain = line;
		char * pass = strstr(domain, "-");
		long offset = (long) pass - (long) domain;
		domain[offset - 2] = 0;
		pass += 3;

		int domainLen = strlen(domain);

		//printf("[%s]->[%s]\n", domain, pass);
		path * head = NULL;
		path * tail = NULL;
		char previous = domain[domainLen - 1];
		int direction = domainLen % 2 == 1 ? UP : LEFT;
		for (int i = 0; i < domainLen; i++)
			{
				char next = domain[i];
				path * current = allocate(previous, next, &pass[i * 2], direction);
				previous = pass[i*2+1];
				if (head == NULL)
					{
						head = current;
						hipMallocManaged((void **) &head->domain, domainLen + 1);
						strcpy(head->domain, domain);
						hipMallocManaged((void **) &head->pass, strlen(pass) + 1);
						strcpy(head->pass, pass);
					}
				if (tail == NULL)
					{
						tail = head;
					}
				else
					{
						tail->next = current;
						tail = current;
					}
				direction = direction == UP ? LEFT : UP;
			}

		return head;
	}





void printPath(path * p)
	{
		if (p != NULL)
			{
				char dir = p->direction == UP ? 'U' : 'L';
				if (p->domain != NULL)
					{
						printf("[%s]->[%s]\n", p->domain, p->pass);
					}
				int value = (int) p->letters[0];
				if (value > 30)
					{
						printf("[%c]->[%c%c%c]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);
					}
				else
					{
						printf("[%c]->[%c%c%c]%c\n", convertChar(p->letters[0]), convertChar(p->letters[1]), convertChar(p->letters[2]), convertChar(p->letters[3]), dir);
						//printf("[%d]->[%d%d%d]%c\n", p->letters[0], p->letters[1], p->letters[2], p->letters[3], dir);

					}
				printPath(p->next);
			}
	}

__device__ void printGrid(grid * g)
	{
		int n = g->size;
		//printf("X=%d Y=%d %c\n", x, y, g->ok);
		for (int row = 0; row < n; row++)
			{
				for (int col = 0; col < n; col++)
					{
						cell c = g->cells[row][col];
						//printf("[%02d][%02d]%02X ",row, col, c[row * N + col].bitmap);
						int value = c.value;
						char printC = ' ';
						if (value < 0)
							{
								value = c.bitmap;
								printf(" %03X%c", value, printC);
							}
						else
							{
								//
								printC = convert(value);
								value = 0;
								printf("  %c  ", printC);
							}

					}
				printf("\n");
			}
	}
